#include "cupoch/visualization/shader/texture_phong_shader.h"

#include "cupoch/geometry/image.h"
#include "cupoch/geometry/pointcloud.h"
#include "cupoch/geometry/trianglemesh.h"
#include "cupoch/visualization/shader/shader.h"
#include "cupoch/visualization/utility/color_map.h"
#include "cupoch/utility/console.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

using namespace cupoch;
using namespace cupoch::visualization;
using namespace cupoch::visualization::glsl;

namespace {

struct copy_trianglemesh_functor {
    copy_trianglemesh_functor(const Eigen::Vector3f* vertices, const Eigen::Vector3f* vertex_normals,
                              const int* triangles, const Eigen::Vector3f* triangle_normals,
                              const Eigen::Vector2f* triangle_uvs,
                              RenderOption::MeshShadeOption shade_option)
                              : vertices_(vertices), vertex_normals_(vertex_normals),
                                triangles_(triangles), triangle_normals_(triangle_normals),
                                triangle_uvs_(triangle_uvs), shade_option_(shade_option) {};
    const Eigen::Vector3f* vertices_;
    const Eigen::Vector3f* vertex_normals_;
    const int* triangles_;
    const Eigen::Vector3f* triangle_normals_;
    const Eigen::Vector2f* triangle_uvs_;
    const RenderOption::MeshShadeOption shade_option_;
    __device__
    thrust::tuple<Eigen::Vector3f, Eigen::Vector3f, Eigen::Vector2f> operator() (size_t k) const {
        int i = k / 3;
        int vi = triangles_[k];
        if (shade_option_ ==
            RenderOption::MeshShadeOption::FlatShade) {
            return thrust::make_tuple(vertices_[vi], triangle_normals_[i], triangle_uvs_[k]);
        } else {
            return thrust::make_tuple(vertices_[vi], vertex_normals_[vi], triangle_uvs_[k]);
        }
    }
};

}

bool TexturePhongShader::Compile() {
    if (CompileShaders(texture_phong_vertex_shader, NULL,
                       texture_phong_fragment_shader) == false) {
        PrintShaderWarning("Compiling shaders failed.");
        return false;
    }
    vertex_position_ = glGetAttribLocation(program_, "vertex_position");
    vertex_normal_ = glGetAttribLocation(program_, "vertex_normal");
    vertex_uv_ = glGetAttribLocation(program_, "vertex_uv");
    MVP_ = glGetUniformLocation(program_, "MVP");
    V_ = glGetUniformLocation(program_, "V");
    M_ = glGetUniformLocation(program_, "M");
    light_position_world_ =
            glGetUniformLocation(program_, "light_position_world_4");
    light_color_ = glGetUniformLocation(program_, "light_color_4");
    light_diffuse_power_ =
            glGetUniformLocation(program_, "light_diffuse_power_4");
    light_specular_power_ =
            glGetUniformLocation(program_, "light_specular_power_4");
    light_specular_shininess_ =
            glGetUniformLocation(program_, "light_specular_shininess_4");
    light_ambient_ = glGetUniformLocation(program_, "light_ambient");

    diffuse_texture_ = glGetUniformLocation(program_, "diffuse_texture");
    return true;
}

void TexturePhongShader::Release() {
    UnbindGeometry();
    ReleaseProgram();
}

bool TexturePhongShader::BindGeometry(const geometry::Geometry &geometry,
                                      const RenderOption &option,
                                      const ViewControl &view) {
    // If there is already geometry, we first unbind it.
    // We use GL_STATIC_DRAW. When geometry changes, we clear buffers and
    // rebind the geometry. Note that this approach is slow. If the geometry is
    // changing per frame, consider implementing a new ShaderWrapper using
    // GL_STREAM_DRAW, and replace UnbindGeometry() with Buffer Object
    // Streaming mechanisms.
    UnbindGeometry();

    // Prepare data to be passed to GPU
    const size_t num_data_size = GetDataSize(geometry);

    // Create buffers and bind the geometry
    glGenBuffers(1, &vertex_position_buffer_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_position_buffer_);
    glBufferData(GL_ARRAY_BUFFER, num_data_size * sizeof(Eigen::Vector3f), 0, GL_STATIC_DRAW);
    cudaSafeCall(hipGraphicsGLRegisterBuffer(&cuda_graphics_resources_[0], vertex_position_buffer_, cudaGraphicsMapFlagsNone));
    glGenBuffers(1, &vertex_normal_buffer_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_normal_buffer_);
    glBufferData(GL_ARRAY_BUFFER, num_data_size * sizeof(Eigen::Vector3f), 0, GL_STATIC_DRAW);
    cudaSafeCall(hipGraphicsGLRegisterBuffer(&cuda_graphics_resources_[1], vertex_normal_buffer_, cudaGraphicsMapFlagsNone));
    glGenBuffers(1, &vertex_uv_buffer_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_uv_buffer_);
    glBufferData(GL_ARRAY_BUFFER, num_data_size * sizeof(Eigen::Vector2f), 0, GL_STATIC_DRAW);
    cudaSafeCall(hipGraphicsGLRegisterBuffer(&cuda_graphics_resources_[2], vertex_uv_buffer_, cudaGraphicsMapFlagsNone));

    Eigen::Vector3f* raw_points_ptr;
    Eigen::Vector3f* raw_normals_ptr;
    Eigen::Vector2f* raw_uvs_ptr;
    size_t n_bytes;
    cudaSafeCall(hipGraphicsMapResources(3, cuda_graphics_resources_));
    cudaSafeCall(hipGraphicsResourceGetMappedPointer((void **)&raw_points_ptr, &n_bytes, cuda_graphics_resources_[0]));
    cudaSafeCall(hipGraphicsResourceGetMappedPointer((void **)&raw_normals_ptr, &n_bytes, cuda_graphics_resources_[1]));
    cudaSafeCall(hipGraphicsResourceGetMappedPointer((void **)&raw_uvs_ptr, &n_bytes, cuda_graphics_resources_[2]));
    thrust::device_ptr<Eigen::Vector3f> dev_points_ptr = thrust::device_pointer_cast(raw_points_ptr);
    thrust::device_ptr<Eigen::Vector3f> dev_normals_ptr = thrust::device_pointer_cast(raw_normals_ptr);
    thrust::device_ptr<Eigen::Vector2f> dev_uvs_ptr = thrust::device_pointer_cast(raw_uvs_ptr);

    if (PrepareBinding(geometry, option, view, dev_points_ptr, dev_normals_ptr, dev_uvs_ptr) == false) {
        PrintShaderWarning("Binding failed when preparing data.");
        return false;
    }
    Unmap(3);
    bound_ = true;
    return true;
}

bool TexturePhongShader::RenderGeometry(const geometry::Geometry &geometry,
                                        const RenderOption &option,
                                        const ViewControl &view) {
    if (PrepareRendering(geometry, option, view) == false) {
        PrintShaderWarning("Rendering failed during preparation.");
        return false;
    }
    glUseProgram(program_);
    glUniformMatrix4fv(MVP_, 1, GL_FALSE, view.GetMVPMatrix().data());
    glUniformMatrix4fv(V_, 1, GL_FALSE, view.GetViewMatrix().data());
    glUniformMatrix4fv(M_, 1, GL_FALSE, view.GetModelMatrix().data());
    glUniformMatrix4fv(light_position_world_, 1, GL_FALSE,
                       light_position_world_data_.data());
    glUniformMatrix4fv(light_color_, 1, GL_FALSE, light_color_data_.data());
    glUniform4fv(light_diffuse_power_, 1, light_diffuse_power_data_.data());
    glUniform4fv(light_specular_power_, 1, light_specular_power_data_.data());
    glUniform4fv(light_specular_shininess_, 1,
                 light_specular_shininess_data_.data());
    glUniform4fv(light_ambient_, 1, light_ambient_data_.data());

    glUniform1i(diffuse_texture_, 0);
    glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, diffuse_texture_buffer_);

    glEnableVertexAttribArray(vertex_position_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_position_buffer_);
    glVertexAttribPointer(vertex_position_, 3, GL_FLOAT, GL_FALSE, 0, NULL);

    glEnableVertexAttribArray(vertex_normal_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_normal_buffer_);
    glVertexAttribPointer(vertex_normal_, 3, GL_FLOAT, GL_FALSE, 0, NULL);

    glEnableVertexAttribArray(vertex_uv_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_uv_buffer_);
    glVertexAttribPointer(vertex_uv_, 2, GL_FLOAT, GL_FALSE, 0, NULL);

    glDrawArrays(draw_arrays_mode_, 0, draw_arrays_size_);

    glDisableVertexAttribArray(vertex_position_);
    glDisableVertexAttribArray(vertex_normal_);
    glDisableVertexAttribArray(vertex_uv_);
    return true;
}

void TexturePhongShader::UnbindGeometry() {
    if (bound_) {
        glDeleteBuffers(1, &vertex_position_buffer_);
        glDeleteBuffers(1, &vertex_normal_buffer_);
        glDeleteBuffers(1, &vertex_uv_buffer_);
        glDeleteTextures(1, &diffuse_texture_buffer_);
        bound_ = false;
    }
}

void TexturePhongShader::SetLighting(const ViewControl &view,
                                     const RenderOption &option) {
    const auto &box = view.GetBoundingBox();
    light_position_world_data_.setOnes();
    light_color_data_.setOnes();
    for (int i = 0; i < 4; i++) {
        light_position_world_data_.block<3, 1>(0, i) =
                box.GetCenter().cast<GLfloat>() +
                (float)box.GetMaxExtent() *
                        ((float)option.light_position_relative_[i](0) *
                                 view.GetRight() +
                         (float)option.light_position_relative_[i](1) *
                                 view.GetUp() +
                         (float)option.light_position_relative_[i](2) *
                                 view.GetFront());
        light_color_data_.block<3, 1>(0, i) =
                option.light_color_[i].cast<GLfloat>();
    }
    if (option.light_on_) {
        light_diffuse_power_data_ =
                Eigen::Vector4f(option.light_diffuse_power_).cast<GLfloat>();
        light_specular_power_data_ =
                Eigen::Vector4f(option.light_specular_power_).cast<GLfloat>();
        light_specular_shininess_data_ =
                Eigen::Vector4f(option.light_specular_shininess_)
                        .cast<GLfloat>();
        light_ambient_data_.block<3, 1>(0, 0) =
                option.light_ambient_color_.cast<GLfloat>();
        light_ambient_data_(3) = 1.0f;
    } else {
        light_diffuse_power_data_ = gl_helper::GLVector4f::Zero();
        light_specular_power_data_ = gl_helper::GLVector4f::Zero();
        light_specular_shininess_data_ = gl_helper::GLVector4f::Ones();
        light_ambient_data_ = gl_helper::GLVector4f(1.0f, 1.0f, 1.0f, 1.0f);
    }
}

bool TexturePhongShaderForTriangleMesh::PrepareRendering(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view) {
    if (geometry.GetGeometryType() !=
                geometry::Geometry::GeometryType::TriangleMesh) {
        PrintShaderWarning("Rendering type is not geometry::TriangleMesh.");
        return false;
    }
    if (option.mesh_show_back_face_) {
        glDisable(GL_CULL_FACE);
    } else {
        glEnable(GL_CULL_FACE);
    }
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GLenum(option.GetGLDepthFunc()));
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
    if (option.mesh_show_wireframe_) {
        glEnable(GL_POLYGON_OFFSET_FILL);
        glPolygonOffset(1.0, 1.0);
    } else {
        glDisable(GL_POLYGON_OFFSET_FILL);
    }
    SetLighting(view, option);
    return true;
}

bool TexturePhongShaderForTriangleMesh::PrepareBinding(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view,
        thrust::device_ptr<Eigen::Vector3f> &points,
        thrust::device_ptr<Eigen::Vector3f> &normals,
        thrust::device_ptr<Eigen::Vector2f> &uvs) {
    if (geometry.GetGeometryType() !=
                geometry::Geometry::GeometryType::TriangleMesh) {
        PrintShaderWarning("Rendering type is not geometry::TriangleMesh.");
        return false;
    }
    const geometry::TriangleMesh &mesh =
            (const geometry::TriangleMesh &)geometry;
    if (mesh.HasTriangles() == false) {
        PrintShaderWarning("Binding failed with empty triangle mesh.");
        return false;
    }
    if (mesh.HasTriangleNormals() == false ||
        mesh.HasVertexNormals() == false) {
        PrintShaderWarning("Binding failed because mesh has no normals.");
        PrintShaderWarning("Call ComputeVertexNormals() before binding.");
        return false;
    }
    copy_trianglemesh_functor func(thrust::raw_pointer_cast(mesh.vertices_.data()),
                                   thrust::raw_pointer_cast(mesh.vertex_normals_.data()),
                                   (int*)(thrust::raw_pointer_cast(mesh.triangles_.data())),
                                   thrust::raw_pointer_cast(mesh.triangle_normals_.data()),
                                   thrust::raw_pointer_cast(mesh.triangle_uvs_.data()),
                                   option.mesh_shade_option_);
    thrust::transform(thrust::make_counting_iterator<size_t>(0),
                      thrust::make_counting_iterator<size_t>(mesh.triangles_.size() * 3),
                      make_tuple_iterator(points, normals, uvs), func);

    glGenTextures(1, &diffuse_texture_);
    glBindTexture(GL_TEXTURE_2D, diffuse_texture_buffer_);

    GLenum format;
    switch (mesh.texture_.num_of_channels_) {
        case 1: {
            format = GL_RED;
            break;
        }
        case 3: {
            format = GL_RGB;
            break;
        }
        case 4: {
            format = GL_RGBA;
            break;
        }
        default: {
            utility::LogWarning("Unknown format, abort!");
            return false;
        }
    }

    GLenum type;
    switch (mesh.texture_.bytes_per_channel_) {
        case 1: {
            type = GL_UNSIGNED_BYTE;
            break;
        }
        case 2: {
            type = GL_UNSIGNED_SHORT;
            break;
        }
        case 4: {
            type = GL_FLOAT;
            break;
        }
        default: {
            utility::LogWarning("Unknown format, abort!");
            return false;
        }
    }
    glTexImage2D(GL_TEXTURE_2D, 0, format, mesh.texture_.width_,
                 mesh.texture_.height_, 0, format, type,
                 thrust::raw_pointer_cast(mesh.texture_.data_.data()));

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_BORDER);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_BORDER);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    draw_arrays_mode_ = GL_TRIANGLES;
    draw_arrays_size_ = GLsizei(mesh.triangles_.size() * 3);
    return true;
}

size_t TexturePhongShaderForTriangleMesh::GetDataSize(const geometry::Geometry &geometry) const {
    return ((const geometry::TriangleMesh &)geometry).triangles_.size() * 3;
}