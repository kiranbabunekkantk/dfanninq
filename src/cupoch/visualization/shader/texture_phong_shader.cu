/**
 * Copyright (c) 2020 Neka-Nat
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 * 
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
**/
#include <hip/hip_runtime.h>

#include "cupoch/geometry/image.h"
#include "cupoch/geometry/pointcloud.h"
#include "cupoch/geometry/trianglemesh.h"
#include "cupoch/utility/console.h"
#include "cupoch/utility/platform.h"
#include "cupoch/visualization/shader/shader.h"
#include "cupoch/visualization/shader/texture_phong_shader.h"
#include "cupoch/visualization/utility/color_map.h"

using namespace cupoch;
using namespace cupoch::visualization;
using namespace cupoch::visualization::glsl;

namespace {

GLenum GetFormat(const geometry::Geometry &geometry) {
    auto it = gl_helper::texture_format_map_.find(
            ((const geometry::TriangleMesh &)geometry)
                    .texture_.num_of_channels_);
    if (it == gl_helper::texture_format_map_.end()) {
        utility::LogWarning("Unknown texture format, abort!");
        return false;
    }
    return it->second;
}

GLenum GetType(const geometry::Geometry &geometry) {
    auto it = gl_helper::texture_type_map_.find(
            ((const geometry::TriangleMesh &)geometry)
                    .texture_.bytes_per_channel_);
    if (it == gl_helper::texture_type_map_.end()) {
        utility::LogWarning("Unknown texture type, abort!");
        return false;
    }
    return it->second;
}

struct copy_trianglemesh_functor {
    copy_trianglemesh_functor(const Eigen::Vector3f *vertices,
                              const Eigen::Vector3f *vertex_normals,
                              const int *triangles,
                              const Eigen::Vector3f *triangle_normals,
                              const Eigen::Vector2f *triangle_uvs,
                              RenderOption::MeshShadeOption shade_option)
        : vertices_(vertices),
          vertex_normals_(vertex_normals),
          triangles_(triangles),
          triangle_normals_(triangle_normals),
          triangle_uvs_(triangle_uvs),
          shade_option_(shade_option){};
    const Eigen::Vector3f *vertices_;
    const Eigen::Vector3f *vertex_normals_;
    const int *triangles_;
    const Eigen::Vector3f *triangle_normals_;
    const Eigen::Vector2f *triangle_uvs_;
    const RenderOption::MeshShadeOption shade_option_;
    __device__ thrust::tuple<Eigen::Vector3f, Eigen::Vector3f, Eigen::Vector2f>
    operator()(size_t k) const {
        int i = k / 3;
        int vi = triangles_[k];
        if (shade_option_ == RenderOption::MeshShadeOption::FlatShade) {
            return thrust::make_tuple(vertices_[vi], triangle_normals_[i],
                                      triangle_uvs_[k]);
        } else {
            return thrust::make_tuple(vertices_[vi], vertex_normals_[vi],
                                      triangle_uvs_[k]);
        }
    }
};

}  // namespace

bool TexturePhongShader::Compile() {
    if (CompileShaders(texture_phong_vertex_shader, NULL,
                       texture_phong_fragment_shader) == false) {
        PrintShaderWarning("Compiling shaders failed.");
        return false;
    }
    vertex_position_ = glGetAttribLocation(program_, "vertex_position");
    vertex_normal_ = glGetAttribLocation(program_, "vertex_normal");
    vertex_uv_ = glGetAttribLocation(program_, "vertex_uv");
    MVP_ = glGetUniformLocation(program_, "MVP");
    V_ = glGetUniformLocation(program_, "V");
    M_ = glGetUniformLocation(program_, "M");
    light_position_world_ =
            glGetUniformLocation(program_, "light_position_world_4");
    light_color_ = glGetUniformLocation(program_, "light_color_4");
    light_diffuse_power_ =
            glGetUniformLocation(program_, "light_diffuse_power_4");
    light_specular_power_ =
            glGetUniformLocation(program_, "light_specular_power_4");
    light_specular_shininess_ =
            glGetUniformLocation(program_, "light_specular_shininess_4");
    light_ambient_ = glGetUniformLocation(program_, "light_ambient");

    diffuse_texture_ = glGetUniformLocation(program_, "diffuse_texture");
    return true;
}

void TexturePhongShader::Release() {
    UnbindGeometry(true);
    ReleaseProgram();
}

bool TexturePhongShader::BindGeometry(const geometry::Geometry &geometry,
                                      const RenderOption &option,
                                      const ViewControl &view) {
    // If there is already geometry, we first unbind it.
    // We use GL_STATIC_DRAW. When geometry changes, we clear buffers and
    // rebind the geometry. Note that this approach is slow. If the geometry is
    // changing per frame, consider implementing a new ShaderWrapper using
    // GL_STREAM_DRAW, and replace UnbindGeometry() with Buffer Object
    // Streaming mechanisms.
    UnbindGeometry();

    // Prepare data to be passed to GPU
    const size_t num_data_size = GetDataSize(geometry);
    const size_t num_texture_height = GetTextureHeight(geometry);
    const size_t num_texture_width = GetTextureWidth(geometry);

    glGenTextures(1, &diffuse_texture_buffer_);
    glBindTexture(GL_TEXTURE_2D, diffuse_texture_buffer_);

    GLenum format = GetFormat(geometry);
    GLenum type = GetType(geometry);
    glTexImage2D(GL_TEXTURE_2D, 0, format, num_texture_width,
                 num_texture_height, 0, format, type, 0);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_BORDER);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_BORDER);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    // Create buffers and bind the geometry
    glGenBuffers(1, &vertex_position_buffer_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_position_buffer_);
    glBufferData(GL_ARRAY_BUFFER, num_data_size * sizeof(Eigen::Vector3f), 0,
                 GL_STATIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    cudaSafeCall(hipGraphicsGLRegisterBuffer(&cuda_graphics_resources_[0],
                                              vertex_position_buffer_,
                                              cudaGraphicsMapFlagsNone));
    glGenBuffers(1, &vertex_normal_buffer_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_normal_buffer_);
    glBufferData(GL_ARRAY_BUFFER, num_data_size * sizeof(Eigen::Vector3f), 0,
                 GL_STATIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    cudaSafeCall(hipGraphicsGLRegisterBuffer(&cuda_graphics_resources_[1],
                                              vertex_normal_buffer_,
                                              cudaGraphicsMapFlagsNone));
    glGenBuffers(1, &vertex_uv_buffer_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_uv_buffer_);
    glBufferData(GL_ARRAY_BUFFER, num_data_size * sizeof(Eigen::Vector2f), 0,
                 GL_STATIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    cudaSafeCall(hipGraphicsGLRegisterBuffer(&cuda_graphics_resources_[2],
                                              vertex_uv_buffer_,
                                              cudaGraphicsMapFlagsNone));
    glGenBuffers(1, &diffuse_texture_pixel_buffer_);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, diffuse_texture_pixel_buffer_);
    size_t texture_size = GetTextureSize(geometry);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, texture_size, 0, GL_STATIC_DRAW);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
    cudaSafeCall(hipGraphicsGLRegisterBuffer(&cuda_graphics_resources_[3],
                                              diffuse_texture_pixel_buffer_,
                                              cudaGraphicsMapFlagsNone));

    Eigen::Vector3f *raw_points_ptr;
    Eigen::Vector3f *raw_normals_ptr;
    Eigen::Vector2f *raw_uvs_ptr;
    uint8_t *raw_render_texture_ptr;
    size_t n_bytes;
    cudaSafeCall(hipGraphicsMapResources(4, cuda_graphics_resources_));
    cudaSafeCall(hipGraphicsResourceGetMappedPointer(
            (void **)&raw_points_ptr, &n_bytes, cuda_graphics_resources_[0]));
    cudaSafeCall(hipGraphicsResourceGetMappedPointer(
            (void **)&raw_normals_ptr, &n_bytes, cuda_graphics_resources_[1]));
    cudaSafeCall(hipGraphicsResourceGetMappedPointer(
            (void **)&raw_uvs_ptr, &n_bytes, cuda_graphics_resources_[2]));
    cudaSafeCall(hipGraphicsResourceGetMappedPointer(
            (void **)&raw_render_texture_ptr, &n_bytes,
            cuda_graphics_resources_[3]));
    thrust::device_ptr<Eigen::Vector3f> dev_points_ptr =
            thrust::device_pointer_cast(raw_points_ptr);
    thrust::device_ptr<Eigen::Vector3f> dev_normals_ptr =
            thrust::device_pointer_cast(raw_normals_ptr);
    thrust::device_ptr<Eigen::Vector2f> dev_uvs_ptr =
            thrust::device_pointer_cast(raw_uvs_ptr);
    thrust::device_ptr<uint8_t> dev_texture_ptr =
            thrust::device_pointer_cast(raw_render_texture_ptr);

    if (PrepareBinding(geometry, option, view, dev_points_ptr, dev_normals_ptr,
                       dev_uvs_ptr, dev_texture_ptr) == false) {
        PrintShaderWarning("Binding failed when preparing data.");
        return false;
    }
    Unmap(4);
    bound_ = true;
    return true;
}

bool TexturePhongShader::RenderGeometry(const geometry::Geometry &geometry,
                                        const RenderOption &option,
                                        const ViewControl &view) {
    if (PrepareRendering(geometry, option, view) == false) {
        PrintShaderWarning("Rendering failed during preparation.");
        return false;
    }

    const size_t num_data_height = GetTextureHeight(geometry);
    const size_t num_data_width = GetTextureWidth(geometry);
    GLenum format = GetFormat(geometry);
    GLenum type = GetType(geometry);

    glUseProgram(program_);
    glUniformMatrix4fv(MVP_, 1, GL_FALSE, view.GetMVPMatrix().data());
    glUniformMatrix4fv(V_, 1, GL_FALSE, view.GetViewMatrix().data());
    glUniformMatrix4fv(M_, 1, GL_FALSE, view.GetModelMatrix().data());
    glUniformMatrix4fv(light_position_world_, 1, GL_FALSE,
                       light_position_world_data_.data());
    glUniformMatrix4fv(light_color_, 1, GL_FALSE, light_color_data_.data());
    glUniform4fv(light_diffuse_power_, 1, light_diffuse_power_data_.data());
    glUniform4fv(light_specular_power_, 1, light_specular_power_data_.data());
    glUniform4fv(light_specular_shininess_, 1,
                 light_specular_shininess_data_.data());
    glUniform4fv(light_ambient_, 1, light_ambient_data_.data());

    glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, diffuse_texture_buffer_);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, diffuse_texture_pixel_buffer_);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, num_data_width, num_data_height,
                    format, type, 0);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
    glUniform1i(diffuse_texture_, 0);

    glEnableVertexAttribArray(vertex_position_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_position_buffer_);
    glVertexAttribPointer(vertex_position_, 3, GL_FLOAT, GL_FALSE, 0, NULL);

    glEnableVertexAttribArray(vertex_normal_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_normal_buffer_);
    glVertexAttribPointer(vertex_normal_, 3, GL_FLOAT, GL_FALSE, 0, NULL);

    glEnableVertexAttribArray(vertex_uv_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_uv_buffer_);
    glVertexAttribPointer(vertex_uv_, 2, GL_FLOAT, GL_FALSE, 0, NULL);

    glDrawArrays(draw_arrays_mode_, 0, draw_arrays_size_);

    glDisableVertexAttribArray(vertex_position_);
    glDisableVertexAttribArray(vertex_normal_);
    glDisableVertexAttribArray(vertex_uv_);
    return true;
}

void TexturePhongShader::UnbindGeometry(bool finalize) {
    if (bound_) {
        if (!finalize) {
            cudaSafeCall(hipGraphicsUnregisterResource(
                    cuda_graphics_resources_[0]));
            cudaSafeCall(hipGraphicsUnregisterResource(
                    cuda_graphics_resources_[1]));
            cudaSafeCall(hipGraphicsUnregisterResource(
                    cuda_graphics_resources_[2]));
            cudaSafeCall(hipGraphicsUnregisterResource(
                    cuda_graphics_resources_[3]));
        }
        glDeleteBuffers(1, &diffuse_texture_buffer_);
        glDeleteBuffers(1, &vertex_position_buffer_);
        glDeleteBuffers(1, &vertex_normal_buffer_);
        glDeleteBuffers(1, &vertex_uv_buffer_);
        glDeleteTextures(1, &diffuse_texture_buffer_);
        bound_ = false;
    }
}

void TexturePhongShader::SetLighting(const ViewControl &view,
                                     const RenderOption &option) {
    const auto &box = view.GetBoundingBox();
    light_position_world_data_.setOnes();
    light_color_data_.setOnes();
    for (int i = 0; i < 4; i++) {
        light_position_world_data_.block<3, 1>(0, i) =
                box.GetCenter().cast<GLfloat>() +
                (float)box.GetMaxExtent() *
                        ((float)option.light_position_relative_[i](0) *
                                 view.GetRight() +
                         (float)option.light_position_relative_[i](1) *
                                 view.GetUp() +
                         (float)option.light_position_relative_[i](2) *
                                 view.GetFront());
        light_color_data_.block<3, 1>(0, i) =
                option.light_color_[i].cast<GLfloat>();
    }
    if (option.light_on_) {
        light_diffuse_power_data_ =
                Eigen::Vector4f(option.light_diffuse_power_).cast<GLfloat>();
        light_specular_power_data_ =
                Eigen::Vector4f(option.light_specular_power_).cast<GLfloat>();
        light_specular_shininess_data_ =
                Eigen::Vector4f(option.light_specular_shininess_)
                        .cast<GLfloat>();
        light_ambient_data_.block<3, 1>(0, 0) =
                option.light_ambient_color_.cast<GLfloat>();
        light_ambient_data_(3) = 1.0f;
    } else {
        light_diffuse_power_data_ = gl_helper::GLVector4f::Zero();
        light_specular_power_data_ = gl_helper::GLVector4f::Zero();
        light_specular_shininess_data_ = gl_helper::GLVector4f::Ones();
        light_ambient_data_ = gl_helper::GLVector4f(1.0f, 1.0f, 1.0f, 1.0f);
    }
}

bool TexturePhongShaderForTriangleMesh::PrepareRendering(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::TriangleMesh) {
        PrintShaderWarning("Rendering type is not geometry::TriangleMesh.");
        return false;
    }
    if (option.mesh_show_back_face_) {
        glDisable(GL_CULL_FACE);
    } else {
        glEnable(GL_CULL_FACE);
    }
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GLenum(option.GetGLDepthFunc()));
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
    if (option.mesh_show_wireframe_) {
        glEnable(GL_POLYGON_OFFSET_FILL);
        glPolygonOffset(1.0, 1.0);
    } else {
        glDisable(GL_POLYGON_OFFSET_FILL);
    }
    SetLighting(view, option);
    return true;
}

bool TexturePhongShaderForTriangleMesh::PrepareBinding(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view,
        thrust::device_ptr<Eigen::Vector3f> &points,
        thrust::device_ptr<Eigen::Vector3f> &normals,
        thrust::device_ptr<Eigen::Vector2f> &uvs,
        thrust::device_ptr<uint8_t> &texture_image) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::TriangleMesh) {
        PrintShaderWarning("Rendering type is not geometry::TriangleMesh.");
        return false;
    }
    const geometry::TriangleMesh &mesh =
            (const geometry::TriangleMesh &)geometry;
    if (mesh.HasTriangles() == false) {
        PrintShaderWarning("Binding failed with empty triangle mesh.");
        return false;
    }
    if (mesh.HasTriangleNormals() == false ||
        mesh.HasVertexNormals() == false) {
        PrintShaderWarning("Binding failed because mesh has no normals.");
        PrintShaderWarning("Call ComputeVertexNormals() before binding.");
        return false;
    }
    copy_trianglemesh_functor func(
            thrust::raw_pointer_cast(mesh.vertices_.data()),
            thrust::raw_pointer_cast(mesh.vertex_normals_.data()),
            (int *)(thrust::raw_pointer_cast(mesh.triangles_.data())),
            thrust::raw_pointer_cast(mesh.triangle_normals_.data()),
            thrust::raw_pointer_cast(mesh.triangle_uvs_.data()),
            option.mesh_shade_option_);
    thrust::transform(
            thrust::make_counting_iterator<size_t>(0),
            thrust::make_counting_iterator<size_t>(mesh.triangles_.size() * 3),
            make_tuple_iterator(points, normals, uvs), func);
    thrust::copy(mesh.texture_.data_.begin(), mesh.texture_.data_.end(),
                 texture_image);

    draw_arrays_mode_ = GL_TRIANGLES;
    draw_arrays_size_ = GLsizei(mesh.triangles_.size() * 3);
    return true;
}

size_t TexturePhongShaderForTriangleMesh::GetDataSize(
        const geometry::Geometry &geometry) const {
    return ((const geometry::TriangleMesh &)geometry).triangles_.size() * 3;
}

size_t TexturePhongShaderForTriangleMesh::GetTextureSize(
        const geometry::Geometry &geometry) const {
    return ((const geometry::TriangleMesh &)geometry).texture_.data_.size();
}

size_t TexturePhongShaderForTriangleMesh::GetTextureHeight(
        const geometry::Geometry &geometry) const {
    return ((const geometry::TriangleMesh &)geometry).texture_.height_;
}

size_t TexturePhongShaderForTriangleMesh::GetTextureWidth(
        const geometry::Geometry &geometry) const {
    return ((const geometry::TriangleMesh &)geometry).texture_.width_;
}