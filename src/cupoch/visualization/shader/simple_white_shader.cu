/**
 * Copyright (c) 2020 Neka-Nat
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 * 
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
**/
#include <hip/hip_runtime.h>

#include "cupoch/geometry/pointcloud.h"
#include "cupoch/geometry/trianglemesh.h"
#include "cupoch/utility/platform.h"
#include "cupoch/visualization/shader/shader.h"
#include "cupoch/visualization/shader/simple_white_shader.h"
#include "cupoch/visualization/utility/color_map.h"

using namespace cupoch;
using namespace cupoch::visualization;
using namespace cupoch::visualization::glsl;

namespace {

struct copy_pointcloud_normal_functor {
    copy_pointcloud_normal_functor(const Eigen::Vector3f *points,
                                   const Eigen::Vector3f *normals,
                                   float line_length)
        : points_(points), normals_(normals), line_length_(line_length){};
    const Eigen::Vector3f *points_;
    const Eigen::Vector3f *normals_;
    const float line_length_;
    __device__ Eigen::Vector3f operator()(size_t idx) {
        int i = idx / 2;
        int j = idx % 2;
        if (j == 0) {
            return points_[i];
        } else {
            return points_[i] + normals_[i] * line_length_;
        }
    }
};

struct copy_mesh_wireflame_functor {
    copy_mesh_wireflame_functor(const Eigen::Vector3f *vertices,
                                const int *triangles)
        : vertices_(vertices), triangles_(triangles){};
    const Eigen::Vector3f *vertices_;
    const int *triangles_;
    __device__ Eigen::Vector3f operator()(size_t k) {
        int vi = triangles_[k];
        return vertices_[vi];
    }
};

}  // namespace

bool SimpleWhiteShader::Compile() {
    if (CompileShaders(simple_white_vertex_shader, NULL,
                       simple_white_fragment_shader) == false) {
        PrintShaderWarning("Compiling shaders failed.");
        return false;
    }
    vertex_position_ = glGetAttribLocation(program_, "vertex_position");
    MVP_ = glGetUniformLocation(program_, "MVP");
    return true;
}

void SimpleWhiteShader::Release() {
    UnbindGeometry(true);
    ReleaseProgram();
}

bool SimpleWhiteShader::BindGeometry(const geometry::Geometry &geometry,
                                     const RenderOption &option,
                                     const ViewControl &view) {
    // If there is already geometry, we first unbind it.
    // We use GL_STATIC_DRAW. When geometry changes, we clear buffers and
    // rebind the geometry. Note that this approach is slow. If the geometry is
    // changing per frame, consider implementing a new ShaderWrapper using
    // GL_STREAM_DRAW, and replace UnbindGeometry() with Buffer Object
    // Streaming mechanisms.
    UnbindGeometry();

    // Prepare data to be passed to GPU
    const size_t num_data_size = GetDataSize(geometry);

    // Create buffers and bind the geometry
    glGenBuffers(1, &vertex_position_buffer_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_position_buffer_);
    glBufferData(GL_ARRAY_BUFFER, num_data_size * sizeof(Eigen::Vector3f), 0,
                 GL_STATIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    cudaSafeCall(hipGraphicsGLRegisterBuffer(&cuda_graphics_resources_[0],
                                              vertex_position_buffer_,
                                              cudaGraphicsMapFlagsNone));

    Eigen::Vector3f *raw_points_ptr;
    size_t n_bytes;
    cudaSafeCall(hipGraphicsMapResources(1, cuda_graphics_resources_));
    cudaSafeCall(hipGraphicsResourceGetMappedPointer(
            (void **)&raw_points_ptr, &n_bytes, cuda_graphics_resources_[0]));
    thrust::device_ptr<Eigen::Vector3f> dev_points_ptr =
            thrust::device_pointer_cast(raw_points_ptr);

    if (PrepareBinding(geometry, option, view, dev_points_ptr) == false) {
        PrintShaderWarning("Binding failed when preparing data.");
        return false;
    }

    Unmap(1);
    bound_ = true;
    return true;
}

bool SimpleWhiteShader::RenderGeometry(const geometry::Geometry &geometry,
                                       const RenderOption &option,
                                       const ViewControl &view) {
    if (PrepareRendering(geometry, option, view) == false) {
        PrintShaderWarning("Rendering failed during preparation.");
        return false;
    }
    glUseProgram(program_);
    glUniformMatrix4fv(MVP_, 1, GL_FALSE, view.GetMVPMatrix().data());
    glEnableVertexAttribArray(vertex_position_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_position_buffer_);
    glVertexAttribPointer(vertex_position_, 3, GL_FLOAT, GL_FALSE, 0, NULL);
    glDrawArrays(draw_arrays_mode_, 0, draw_arrays_size_);
    glDisableVertexAttribArray(vertex_position_);
    return true;
}

void SimpleWhiteShader::UnbindGeometry(bool finalize) {
    if (bound_) {
        if (!finalize)
            cudaSafeCall(hipGraphicsUnregisterResource(
                    cuda_graphics_resources_[0]));
        glDeleteBuffers(1, &vertex_position_buffer_);
        bound_ = false;
    }
}

bool SimpleWhiteShaderForPointCloudNormal::PrepareRendering(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::PointCloud) {
        PrintShaderWarning("Rendering type is not geometry::PointCloud.");
        return false;
    }
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GLenum(option.GetGLDepthFunc()));
    return true;
}

bool SimpleWhiteShaderForPointCloudNormal::PrepareBinding(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view,
        thrust::device_ptr<Eigen::Vector3f> &points) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::PointCloud) {
        PrintShaderWarning("Rendering type is not geometry::PointCloud.");
        return false;
    }
    const geometry::PointCloud &pointcloud =
            (const geometry::PointCloud &)geometry;
    if (pointcloud.HasPoints() == false) {
        PrintShaderWarning("Binding failed with empty pointcloud.");
        return false;
    }
    float line_length =
            option.point_size_ * 0.01 * view.GetBoundingBox().GetMaxExtent();
    copy_pointcloud_normal_functor func(
            thrust::raw_pointer_cast(pointcloud.points_.data()),
            thrust::raw_pointer_cast(pointcloud.normals_.data()), line_length);
    thrust::transform(
            thrust::make_counting_iterator<size_t>(0),
            thrust::make_counting_iterator(pointcloud.points_.size() * 2),
            points, func);
    draw_arrays_mode_ = GL_LINES;
    draw_arrays_size_ = GLsizei(pointcloud.points_.size() * 2);
    return true;
}

size_t SimpleWhiteShaderForPointCloudNormal::GetDataSize(
        const geometry::Geometry &geometry) const {
    return ((const geometry::PointCloud &)geometry).points_.size() * 2;
}

bool SimpleWhiteShaderForTriangleMeshWireFrame::PrepareRendering(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::TriangleMesh) {
        PrintShaderWarning("Rendering type is not geometry::TriangleMesh.");
        return false;
    }
    glLineWidth(1.0f);
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GL_LEQUAL);
    glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
    glDisable(GL_POLYGON_OFFSET_FILL);
    return true;
}

bool SimpleWhiteShaderForTriangleMeshWireFrame::PrepareBinding(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view,
        thrust::device_ptr<Eigen::Vector3f> &points) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::TriangleMesh) {
        PrintShaderWarning("Rendering type is not geometry::TriangleMesh.");
        return false;
    }
    const geometry::TriangleMesh &mesh =
            (const geometry::TriangleMesh &)geometry;
    if (mesh.HasTriangles() == false) {
        PrintShaderWarning("Binding failed with empty geometry::TriangleMesh.");
        return false;
    }
    copy_mesh_wireflame_functor func(
            thrust::raw_pointer_cast(mesh.vertices_.data()),
            (int *)(thrust::raw_pointer_cast(mesh.triangles_.data())));
    thrust::transform(
            thrust::make_counting_iterator<size_t>(0),
            thrust::make_counting_iterator<size_t>(mesh.triangles_.size() * 3),
            points, func);
    draw_arrays_mode_ = GL_TRIANGLES;
    draw_arrays_size_ = GLsizei(mesh.triangles_.size() * 3);
    return true;
}

size_t SimpleWhiteShaderForTriangleMeshWireFrame::GetDataSize(
        const geometry::Geometry &geometry) const {
    return ((const geometry::TriangleMesh &)geometry).triangles_.size() * 3;
}