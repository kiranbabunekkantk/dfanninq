#include "cupoch/visualization/shader/texture_simple_shader.h"

#include "cupoch/geometry/trianglemesh.h"
#include "cupoch/visualization/shader/shader.h"
#include "cupoch/visualization/utility/color_map.h"
#include "cupoch/utility/console.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

using namespace cupoch;
using namespace cupoch::visualization;
using namespace cupoch::visualization::glsl;

namespace {

struct copy_trianglemesh_functor {
    copy_trianglemesh_functor(const Eigen::Vector3f* vertices, const int* triangles,
                              const Eigen::Vector2f* triangle_uvs)
                              : vertices_(vertices), triangles_(triangles), triangle_uvs_(triangle_uvs) {};
    const Eigen::Vector3f* vertices_;
    const int* triangles_;
    const Eigen::Vector2f* triangle_uvs_;
    __device__
    thrust::tuple<Eigen::Vector3f, Eigen::Vector2f> operator() (size_t k) const {
        int vi = triangles_[k];
        return thrust::make_tuple(vertices_[vi], triangle_uvs_[k]);
    }
};

}

bool TextureSimpleShader::Compile() {
    if (CompileShaders(texture_simple_vertex_shader, NULL,
                       texture_simple_fragment_shader) == false) {
        PrintShaderWarning("Compiling shaders failed.");
        return false;
    }
    vertex_position_ = glGetAttribLocation(program_, "vertex_position");
    vertex_uv_ = glGetAttribLocation(program_, "vertex_uv");
    texture_ = glGetUniformLocation(program_, "diffuse_texture");
    MVP_ = glGetUniformLocation(program_, "MVP");
    return true;
}

void TextureSimpleShader::Release() {
    UnbindGeometry();
    ReleaseProgram();
}

bool TextureSimpleShader::BindGeometry(const geometry::Geometry &geometry,
                                       const RenderOption &option,
                                       const ViewControl &view) {
    // If there is already geometry, we first unbind it.
    // We use GL_STATIC_DRAW. When geometry changes, we clear buffers and
    // rebind the geometry. Note that this approach is slow. If the geometry is
    // changing per frame, consider implementing a new ShaderWrapper using
    // GL_STREAM_DRAW, and replace InvalidateGeometry() with Buffer Object
    // Streaming mechanisms.
    UnbindGeometry();

    // Prepare data to be passed to GPU
    const size_t num_data_size = GetDataSize(geometry);

    // Create buffers and bind the geometry
    glGenBuffers(1, &vertex_position_buffer_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_position_buffer_);
    glBufferData(GL_ARRAY_BUFFER, num_data_size * sizeof(Eigen::Vector3f), 0, GL_STATIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    cudaSafeCall(hipGraphicsGLRegisterBuffer(&cuda_graphics_resources_[0], vertex_position_buffer_, cudaGraphicsMapFlagsNone));
    glGenBuffers(1, &vertex_uv_buffer_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_uv_buffer_);
    glBufferData(GL_ARRAY_BUFFER, num_data_size * sizeof(Eigen::Vector2f), 0, GL_STATIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    cudaSafeCall(hipGraphicsGLRegisterBuffer(&cuda_graphics_resources_[1], vertex_uv_buffer_, cudaGraphicsMapFlagsNone));

    Eigen::Vector3f* raw_points_ptr;
    Eigen::Vector2f* raw_uvs_ptr;
    size_t n_bytes;
    cudaSafeCall(hipGraphicsMapResources(2, cuda_graphics_resources_));
    cudaSafeCall(hipGraphicsResourceGetMappedPointer((void **)&raw_points_ptr, &n_bytes, cuda_graphics_resources_[0]));
    cudaSafeCall(hipGraphicsResourceGetMappedPointer((void **)&raw_uvs_ptr, &n_bytes, cuda_graphics_resources_[1]));
    thrust::device_ptr<Eigen::Vector3f> dev_points_ptr = thrust::device_pointer_cast(raw_points_ptr);
    thrust::device_ptr<Eigen::Vector2f> dev_uvs_ptr = thrust::device_pointer_cast(raw_uvs_ptr);

    if (PrepareBinding(geometry, option, view, dev_points_ptr, dev_uvs_ptr) == false) {
        PrintShaderWarning("Binding failed when preparing data.");
        return false;
    }
    Unmap(2);
    bound_ = true;
    return true;
}

bool TextureSimpleShader::RenderGeometry(const geometry::Geometry &geometry,
                                         const RenderOption &option,
                                         const ViewControl &view) {
    if (PrepareRendering(geometry, option, view) == false) {
        PrintShaderWarning("Rendering failed during preparation.");
        return false;
    }
    glUseProgram(program_);
    glUniformMatrix4fv(MVP_, 1, GL_FALSE, view.GetMVPMatrix().data());

    glUniform1i(texture_, 0);
    glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, texture_buffer_);

    glEnableVertexAttribArray(vertex_position_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_position_buffer_);
    glVertexAttribPointer(vertex_position_, 3, GL_FLOAT, GL_FALSE, 0, NULL);

    glEnableVertexAttribArray(vertex_uv_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_uv_buffer_);
    glVertexAttribPointer(vertex_uv_, 2, GL_FLOAT, GL_FALSE, 0, NULL);

    glDrawArrays(draw_arrays_mode_, 0, draw_arrays_size_);
    glDisableVertexAttribArray(vertex_position_);
    glDisableVertexAttribArray(vertex_uv_);
    return true;
}

void TextureSimpleShader::UnbindGeometry() {
    if (bound_) {
        cudaSafeCall(hipGraphicsUnregisterResource(cuda_graphics_resources_[0]));
        cudaSafeCall(hipGraphicsUnregisterResource(cuda_graphics_resources_[1]));
        glDeleteBuffers(1, &vertex_position_buffer_);
        glDeleteBuffers(1, &vertex_uv_buffer_);
        glDeleteTextures(1, &texture_buffer_);
        bound_ = false;
    }
}

bool TextureSimpleShaderForTriangleMesh::PrepareRendering(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view) {
    if (geometry.GetGeometryType() !=
                geometry::Geometry::GeometryType::TriangleMesh) {
        PrintShaderWarning("Rendering type is not geometry::TriangleMesh.");
        return false;
    }
    if (option.mesh_show_back_face_) {
        glDisable(GL_CULL_FACE);
    } else {
        glEnable(GL_CULL_FACE);
    }
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GLenum(option.GetGLDepthFunc()));
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
    if (option.mesh_show_wireframe_) {
        glEnable(GL_POLYGON_OFFSET_FILL);
        glPolygonOffset(1.0, 1.0);
    } else {
        glDisable(GL_POLYGON_OFFSET_FILL);
    }
    return true;
}

bool TextureSimpleShaderForTriangleMesh::PrepareBinding(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view,
        thrust::device_ptr<Eigen::Vector3f> &points,
        thrust::device_ptr<Eigen::Vector2f> &uvs) {
    if (geometry.GetGeometryType() !=
                geometry::Geometry::GeometryType::TriangleMesh) {
        PrintShaderWarning("Rendering type is not geometry::TriangleMesh.");
        return false;
    }
    const geometry::TriangleMesh &mesh =
            (const geometry::TriangleMesh &)geometry;
    if (mesh.HasTriangles() == false) {
        PrintShaderWarning("Binding failed with empty triangle mesh.");
        return false;
    }
    copy_trianglemesh_functor func(thrust::raw_pointer_cast(mesh.vertices_.data()),
                                   (int*)(thrust::raw_pointer_cast(mesh.triangles_.data())),
                                   thrust::raw_pointer_cast(mesh.triangle_uvs_.data()));
    thrust::transform(thrust::make_counting_iterator<size_t>(0),
                      thrust::make_counting_iterator(mesh.triangles_.size() * 3),
                      make_tuple_iterator(points, uvs), func);

    glGenTextures(1, &texture_);
    glBindTexture(GL_TEXTURE_2D, texture_buffer_);

    GLenum format;
    switch (mesh.texture_.num_of_channels_) {
        case 1: {
            format = GL_RED;
            break;
        }
        case 3: {
            format = GL_RGB;
            break;
        }
        case 4: {
            format = GL_RGBA;
            break;
        }
        default: {
            utility::LogWarning("Unknown format, abort!");
            return false;
        }
    }

    GLenum type;
    switch (mesh.texture_.bytes_per_channel_) {
        case 1: {
            type = GL_UNSIGNED_BYTE;
            break;
        }
        case 2: {
            type = GL_UNSIGNED_SHORT;
            break;
        }
        case 4: {
            type = GL_FLOAT;
            break;
        }
        default: {
            utility::LogWarning("Unknown format, abort!");
            return false;
        }
    }
    glTexImage2D(GL_TEXTURE_2D, 0, format, mesh.texture_.width_,
                 mesh.texture_.height_, 0, format, type,
                 thrust::raw_pointer_cast(mesh.texture_.data_.data()));

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_BORDER);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_BORDER);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    draw_arrays_mode_ = GL_TRIANGLES;
    draw_arrays_size_ = GLsizei(mesh.triangles_.size() * 3);
    return true;
}

size_t TextureSimpleShaderForTriangleMesh::GetDataSize(const geometry::Geometry &geometry) const {
    return ((const geometry::TriangleMesh &)geometry).triangles_.size() * 3;
}