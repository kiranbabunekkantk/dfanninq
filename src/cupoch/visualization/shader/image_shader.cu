#include "hip/hip_runtime.h"
#include "cupoch/visualization/shader/image_shader.h"

#include <algorithm>

#include "cupoch/geometry/image.h"
#include "cupoch/visualization/shader/shader.h"
#include "cupoch/visualization/utility/color_map.h"
#include "cupoch/utility/range.h"

using namespace cupoch;
using namespace cupoch::visualization;
using namespace cupoch::visualization::glsl;

namespace {

__device__
uint8_t ConvertColorFromFloatToUnsignedChar(float color) {
    if (std::isnan(color)) {
        return 0;
    } else {
        thrust::minimum<float> min;
        thrust::maximum<float> max;
        float unified_color = min(1.0f, max(0.0f, color));
        return (uint8_t)(unified_color * 255.0f);
    }
}

struct copy_float_gray_image_functor {
    copy_float_gray_image_functor(const uint8_t* gray) : gray_(gray) {};
    const uint8_t* gray_;
    __device__
    uint8_t operator() (size_t k) const {
        int idx = k / 3;
        float *p = (float *)(gray_ + idx * 4);
        uint8_t color = ConvertColorFromFloatToUnsignedChar(*p);
        return color;
    }
};

struct copy_float_rgb_image_functor {
    copy_float_rgb_image_functor(const uint8_t* rgb) : rgb_(rgb) {};
    const uint8_t* rgb_;
    __device__
    uint8_t operator() (size_t idx) const {
        float *p = (float *)(rgb_ + idx * 4);
        return ConvertColorFromFloatToUnsignedChar(*p);
    }
};

struct copy_int16_rgb_image_functor {
    copy_int16_rgb_image_functor(const uint8_t* rgb) : rgb_(rgb) {};
    const uint8_t* rgb_;
    __device__
    uint8_t operator() (size_t idx) const {
        uint16_t *p = (uint16_t *)(rgb_ + idx * 2);
        return (uint8_t)((*p) & 0xff);
    }
};

struct copy_depth_image_functor {
    copy_depth_image_functor(const uint8_t* depth, int max_depth)
        : depth_(depth), max_depth_(max_depth) {};
    const uint8_t* depth_;
    const int max_depth_;
    const thrust::device_ptr<const ColorMap> global_color_map_ = GetGlobalColorMap();
    __device__
    uint8_t operator() (size_t k) const {
        thrust::minimum<float> min;
        int i = k / 3;
        int j = k % 3;
        uint16_t *p = (uint16_t *)(depth_ + i * 2);
        float depth = min(float(*p) / float(max_depth_), 1.0);
        Eigen::Vector3f color = global_color_map_.get()->GetColor(depth);
        return (uint8_t)(color(j) * 255);
    }
};

}  // unnamed namespace

bool ImageShader::Compile() {
    if (CompileShaders(image_vertex_shader, NULL, image_fragment_shader) == false) {
        PrintShaderWarning("Compiling shaders failed.");
        return false;
    }
    vertex_position_ = glGetAttribLocation(program_, "vertex_position");
    vertex_UV_ = glGetAttribLocation(program_, "vertex_UV");
    image_texture_ = glGetUniformLocation(program_, "image_texture");
    vertex_scale_ = glGetUniformLocation(program_, "vertex_scale");
    return true;
}

void ImageShader::Release() {
    UnbindGeometry();
    ReleaseProgram();
}

bool ImageShader::BindGeometry(const geometry::Geometry &geometry,
                               const RenderOption &option,
                               const ViewControl &view) {
    // If there is already geometry, we first unbind it.
    // We use GL_STATIC_DRAW. When geometry changes, we clear buffers and
    // rebind the geometry. Note that this approach is slow. If the geometry is
    // changing per frame, consider implementing a new ShaderWrapper using
    // GL_STREAM_DRAW, and replace UnbindGeometry() with Buffer Object
    // Streaming mechanisms.
    UnbindGeometry();

    // Prepare data to be passed to GPU
    geometry::Image render_image;
    if (PrepareBinding(geometry, option, view, render_image) == false) {
        PrintShaderWarning("Binding failed when preparing data.");
        return false;
    }

    // Create buffers and bind the geometry
    const GLfloat vertex_position_buffer_data[18] = {
            -1.0f, -1.0f, 0.0f, 1.0f, -1.0f, 0.0f, 1.0f,  1.0f, 0.0f,
            -1.0f, -1.0f, 0.0f, 1.0f, 1.0f,  0.0f, -1.0f, 1.0f, 0.0f,
    };
    const GLfloat vertex_UV_buffer_data[12] = {
            0.0f, 1.0f, 1.0f, 1.0f, 1.0f, 0.0f,
            0.0f, 1.0f, 1.0f, 0.0f, 0.0f, 0.0f,
    };
    glGenBuffers(1, &vertex_position_buffer_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_position_buffer_);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertex_position_buffer_data),
                 vertex_position_buffer_data, GL_STATIC_DRAW);
    glGenBuffers(1, &vertex_UV_buffer_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_UV_buffer_);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertex_UV_buffer_data),
                 vertex_UV_buffer_data, GL_STATIC_DRAW);

    glGenTextures(1, &image_texture_buffer_);
    glBindTexture(GL_TEXTURE_2D, image_texture_buffer_);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, render_image.width_,
                 render_image.height_, 0, GL_RGB, GL_UNSIGNED_BYTE,
                 thrust::raw_pointer_cast(render_image.data_.data()));

    if (option.interpolation_option_ ==
        RenderOption::TextureInterpolationOption::Nearest) {
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    } else {
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER,
                        GL_LINEAR_MIPMAP_LINEAR);
        glGenerateMipmap(GL_TEXTURE_2D);
    }

    bound_ = true;
    return true;
}

bool ImageShader::RenderGeometry(const geometry::Geometry &geometry,
                                 const RenderOption &option,
                                 const ViewControl &view) {
    if (PrepareRendering(geometry, option, view) == false) {
        PrintShaderWarning("Rendering failed during preparation.");
        return false;
    }

    glUseProgram(program_);
    glUniform3fv(vertex_scale_, 1, vertex_scale_data_.data());
    glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, image_texture_buffer_);
    glUniform1i(image_texture_, 0);
    glEnableVertexAttribArray(vertex_position_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_position_buffer_);
    glVertexAttribPointer(vertex_position_, 3, GL_FLOAT, GL_FALSE, 0, NULL);
    glEnableVertexAttribArray(vertex_UV_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_UV_buffer_);
    glVertexAttribPointer(vertex_UV_, 2, GL_FLOAT, GL_FALSE, 0, NULL);
    glDrawArrays(draw_arrays_mode_, 0, draw_arrays_size_);
    glDisableVertexAttribArray(vertex_position_);
    glDisableVertexAttribArray(vertex_UV_);

    return true;
}

void ImageShader::UnbindGeometry() {
    if (bound_) {
        glDeleteBuffers(1, &vertex_position_buffer_);
        glDeleteBuffers(1, &vertex_UV_buffer_);
        glDeleteTextures(1, &image_texture_buffer_);
        bound_ = false;
    }
}

bool ImageShaderForImage::PrepareRendering(const geometry::Geometry &geometry,
                                           const RenderOption &option,
                                           const ViewControl &view) {
    if (geometry.GetGeometryType() != geometry::Geometry::GeometryType::Image) {
        PrintShaderWarning("Rendering type is not geometry::Image.");
        return false;
    }
    const geometry::Image &image = (const geometry::Image &)geometry;
    GLfloat ratio_x, ratio_y;
    switch (option.image_stretch_option_) {
        case RenderOption::ImageStretchOption::StretchKeepRatio:
            ratio_x = GLfloat(image.width_) / GLfloat(view.GetWindowWidth());
            ratio_y = GLfloat(image.height_) / GLfloat(view.GetWindowHeight());
            if (ratio_x < ratio_y) {
                ratio_x /= ratio_y;
                ratio_y = 1.0f;
            } else {
                ratio_y /= ratio_x;
                ratio_x = 1.0f;
            }
            break;
        case RenderOption::ImageStretchOption::StretchWithWindow:
            ratio_x = 1.0f;
            ratio_y = 1.0f;
            break;
        case RenderOption::ImageStretchOption::OriginalSize:
        default:
            ratio_x = GLfloat(image.width_) / GLfloat(view.GetWindowWidth());
            ratio_y = GLfloat(image.height_) / GLfloat(view.GetWindowHeight());
            break;
    }
    vertex_scale_data_(0) = ratio_x;
    vertex_scale_data_(1) = ratio_y;
    vertex_scale_data_(2) = 1.0f;
    glDisable(GL_DEPTH_TEST);
    return true;
}

bool ImageShaderForImage::PrepareBinding(const geometry::Geometry &geometry,
                                         const RenderOption &option,
                                         const ViewControl &view,
                                         geometry::Image &render_image) {
    if (geometry.GetGeometryType() != geometry::Geometry::GeometryType::Image) {
        PrintShaderWarning("Rendering type is not geometry::Image.");
        return false;
    }
    const geometry::Image &image = (const geometry::Image &)geometry;
    if (image.HasData() == false) {
        PrintShaderWarning("Binding failed with empty image.");
        return false;
    }

    if (image.num_of_channels_ == 3 && image.bytes_per_channel_ == 1) {
        render_image = image;
    } else {
        render_image.Prepare(image.width_, image.height_, 3, 1);
        if (image.num_of_channels_ == 1 && image.bytes_per_channel_ == 1) {
            // grayscale image
            thrust::repeated_range<thrust::device_vector<uint8_t>::const_iterator> range(image.data_.begin(), image.data_.end(), 3);
            thrust::copy(range.begin(), range.end(), render_image.data_.begin());
        } else if (image.num_of_channels_ == 1 &&
                   image.bytes_per_channel_ == 4) {
            // grayscale image with floating point per channel
            copy_float_gray_image_functor func(thrust::raw_pointer_cast(image.data_.data()));
            thrust::transform(thrust::make_counting_iterator<size_t>(0),
                              thrust::make_counting_iterator<size_t>(image.height_ * image.width_ * 3),
                              render_image.data_.begin(), func);
        } else if (image.num_of_channels_ == 3 &&
                   image.bytes_per_channel_ == 4) {
            // RGB image with floating point per channel
            copy_float_rgb_image_functor func(thrust::raw_pointer_cast(image.data_.data()));
            thrust::transform(thrust::make_counting_iterator<size_t>(0),
                              thrust::make_counting_iterator<size_t>(image.height_ * image.width_ * 3),
                              render_image.data_.begin(), func);
        } else if (image.num_of_channels_ == 3 &&
                   image.bytes_per_channel_ == 2) {
            // image with RGB channels, each channel is a 16-bit integer
            copy_int16_rgb_image_functor func(thrust::raw_pointer_cast(image.data_.data()));
            thrust::transform(thrust::make_counting_iterator<size_t>(0),
                              thrust::make_counting_iterator<size_t>(image.height_ * image.width_ * 3),
                              render_image.data_.begin(), func);
        } else if (image.num_of_channels_ == 1 &&
                   image.bytes_per_channel_ == 2) {
            // depth image, one channel of 16-bit integer
            const int max_depth = option.image_max_depth_;
            copy_depth_image_functor func(thrust::raw_pointer_cast(image.data_.data()), max_depth);
            thrust::transform(thrust::make_counting_iterator<size_t>(0),
                              thrust::make_counting_iterator<size_t>(image.height_ * image.width_ * 3),
                              render_image.data_.begin(), func);
        }
    }

    draw_arrays_mode_ = GL_TRIANGLES;
    draw_arrays_size_ = 6;
    return true;
}
