/**
 * Copyright (c) 2020 Neka-Nat
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 * 
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
**/
#include <hip/hip_runtime.h>

#include "cupoch/geometry/pointcloud.h"
#include "cupoch/geometry/trianglemesh.h"
#include "cupoch/utility/platform.h"
#include "cupoch/visualization/shader/normal_shader.h"
#include "cupoch/visualization/shader/shader.h"

using namespace cupoch;
using namespace cupoch::visualization;
using namespace cupoch::visualization::glsl;

namespace {

struct copy_trianglemesh_functor {
    copy_trianglemesh_functor(const Eigen::Vector3f *vertices,
                              const Eigen::Vector3f *vertex_normals,
                              const int *triangles,
                              const Eigen::Vector3f *triangle_normals,
                              RenderOption::MeshShadeOption shade_option)
        : vertices_(vertices),
          vertex_normals_(vertex_normals),
          triangles_(triangles),
          triangle_normals_(triangle_normals),
          shade_option_(shade_option){};
    const Eigen::Vector3f *vertices_;
    const Eigen::Vector3f *vertex_normals_;
    const int *triangles_;
    const Eigen::Vector3f *triangle_normals_;
    const RenderOption::MeshShadeOption shade_option_;
    __device__ thrust::tuple<Eigen::Vector3f, Eigen::Vector3f> operator()(
            size_t k) const {
        int i = k / 3;
        int vi = triangles_[k];
        const auto &vertex = vertices_[vi];
        return (shade_option_ == RenderOption::MeshShadeOption::FlatShade)
                       ? thrust::make_tuple(vertex, triangle_normals_[i])
                       : thrust::make_tuple(vertex, vertex_normals_[vi]);
    }
};

}  // namespace

bool NormalShader::Compile() {
    if (CompileShaders(normal_vertex_shader, NULL, normal_fragment_shader) ==
        false) {
        PrintShaderWarning("Compiling shaders failed.");
        return false;
    }
    vertex_position_ = glGetAttribLocation(program_, "vertex_position");
    vertex_normal_ = glGetAttribLocation(program_, "vertex_normal");
    MVP_ = glGetUniformLocation(program_, "MVP");
    V_ = glGetUniformLocation(program_, "V");
    M_ = glGetUniformLocation(program_, "M");
    return true;
}

void NormalShader::Release() {
    UnbindGeometry(true);
    ReleaseProgram();
}

bool NormalShader::BindGeometry(const geometry::Geometry &geometry,
                                const RenderOption &option,
                                const ViewControl &view) {
    // If there is already geometry, we first unbind it.
    // We use GL_STATIC_DRAW. When geometry changes, we clear buffers and
    // rebind the geometry. Note that this approach is slow. If the geometry is
    // changing per frame, consider implementing a new ShaderWrapper using
    // GL_STREAM_DRAW, and replace UnbindGeometry() with Buffer Object
    // Streaming mechanisms.
    UnbindGeometry();

    // Prepare data to be passed to GPU
    const size_t num_data_size = GetDataSize(geometry);

    // Create buffers and bind the geometry
    glGenBuffers(1, &vertex_position_buffer_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_position_buffer_);
    glBufferData(GL_ARRAY_BUFFER, num_data_size * sizeof(Eigen::Vector3f), 0,
                 GL_STATIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    cudaSafeCall(hipGraphicsGLRegisterBuffer(&cuda_graphics_resources_[0],
                                              vertex_position_buffer_,
                                              cudaGraphicsMapFlagsNone));
    glGenBuffers(1, &vertex_normal_buffer_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_normal_buffer_);
    glBufferData(GL_ARRAY_BUFFER, num_data_size * sizeof(Eigen::Vector3f), 0,
                 GL_STATIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    cudaSafeCall(hipGraphicsGLRegisterBuffer(&cuda_graphics_resources_[1],
                                              vertex_normal_buffer_,
                                              cudaGraphicsMapFlagsNone));

    Eigen::Vector3f *raw_points_ptr;
    Eigen::Vector3f *raw_normals_ptr;
    size_t n_bytes;
    cudaSafeCall(hipGraphicsMapResources(2, cuda_graphics_resources_));
    cudaSafeCall(hipGraphicsResourceGetMappedPointer(
            (void **)&raw_points_ptr, &n_bytes, cuda_graphics_resources_[0]));
    cudaSafeCall(hipGraphicsResourceGetMappedPointer(
            (void **)&raw_normals_ptr, &n_bytes, cuda_graphics_resources_[1]));
    thrust::device_ptr<Eigen::Vector3f> dev_points_ptr =
            thrust::device_pointer_cast(raw_points_ptr);
    thrust::device_ptr<Eigen::Vector3f> dev_normals_ptr =
            thrust::device_pointer_cast(raw_normals_ptr);

    if (PrepareBinding(geometry, option, view, dev_points_ptr,
                       dev_normals_ptr) == false) {
        PrintShaderWarning("Binding failed when preparing data.");
        return false;
    }

    Unmap(2);
    bound_ = true;
    return true;
}

bool NormalShader::RenderGeometry(const geometry::Geometry &geometry,
                                  const RenderOption &option,
                                  const ViewControl &view) {
    if (PrepareRendering(geometry, option, view) == false) {
        PrintShaderWarning("Rendering failed during preparation.");
        return false;
    }
    glUseProgram(program_);
    glUniformMatrix4fv(MVP_, 1, GL_FALSE, view.GetMVPMatrix().data());
    glUniformMatrix4fv(V_, 1, GL_FALSE, view.GetViewMatrix().data());
    glUniformMatrix4fv(M_, 1, GL_FALSE, view.GetModelMatrix().data());
    glEnableVertexAttribArray(vertex_position_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_position_buffer_);
    glVertexAttribPointer(vertex_position_, 3, GL_FLOAT, GL_FALSE, 0, NULL);
    glEnableVertexAttribArray(vertex_normal_);
    glBindBuffer(GL_ARRAY_BUFFER, vertex_normal_buffer_);
    glVertexAttribPointer(vertex_normal_, 3, GL_FLOAT, GL_FALSE, 0, NULL);
    glDrawArrays(draw_arrays_mode_, 0, draw_arrays_size_);
    glDisableVertexAttribArray(vertex_position_);
    glDisableVertexAttribArray(vertex_normal_);
    return true;
}

void NormalShader::UnbindGeometry(bool finalize) {
    if (bound_) {
        if (!finalize) {
            cudaSafeCall(hipGraphicsUnregisterResource(
                    cuda_graphics_resources_[0]));
            cudaSafeCall(hipGraphicsUnregisterResource(
                    cuda_graphics_resources_[1]));
        }
        glDeleteBuffers(1, &vertex_position_buffer_);
        glDeleteBuffers(1, &vertex_normal_buffer_);
        bound_ = false;
    }
}

bool NormalShaderForPointCloud::PrepareRendering(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::PointCloud) {
        PrintShaderWarning("Rendering type is not geometry::PointCloud.");
        return false;
    }
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GLenum(option.GetGLDepthFunc()));
    glPointSize(GLfloat(option.point_size_));
    return true;
}

bool NormalShaderForPointCloud::PrepareBinding(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view,
        thrust::device_ptr<Eigen::Vector3f> &points,
        thrust::device_ptr<Eigen::Vector3f> &normals) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::PointCloud) {
        PrintShaderWarning("Rendering type is not geometry::PointCloud.");
        return false;
    }
    const geometry::PointCloud &pointcloud =
            (const geometry::PointCloud &)geometry;
    if (pointcloud.HasPoints() == false) {
        PrintShaderWarning("Binding failed with empty pointcloud.");
        return false;
    }
    if (pointcloud.HasNormals() == false) {
        PrintShaderWarning("Binding failed with pointcloud with no normals.");
        return false;
    }
    thrust::copy(pointcloud.points_.begin(), pointcloud.points_.end(), points);
    thrust::copy(pointcloud.normals_.begin(), pointcloud.normals_.end(),
                 normals);
    draw_arrays_mode_ = GL_POINTS;
    draw_arrays_size_ = GLsizei(pointcloud.points_.size());
    return true;
}

size_t NormalShaderForPointCloud::GetDataSize(
        const geometry::Geometry &geometry) const {
    return ((const geometry::PointCloud &)geometry).points_.size();
}

bool NormalShaderForTriangleMesh::PrepareRendering(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::TriangleMesh) {
        PrintShaderWarning("Rendering type is not geometry::TriangleMesh.");
        return false;
    }
    if (option.mesh_show_back_face_) {
        glDisable(GL_CULL_FACE);
    } else {
        glEnable(GL_CULL_FACE);
    }
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GLenum(option.GetGLDepthFunc()));
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
    if (option.mesh_show_wireframe_) {
        glEnable(GL_POLYGON_OFFSET_FILL);
        glPolygonOffset(1.0, 1.0);
    } else {
        glDisable(GL_POLYGON_OFFSET_FILL);
    }
    return true;
}

bool NormalShaderForTriangleMesh::PrepareBinding(
        const geometry::Geometry &geometry,
        const RenderOption &option,
        const ViewControl &view,
        thrust::device_ptr<Eigen::Vector3f> &points,
        thrust::device_ptr<Eigen::Vector3f> &normals) {
    if (geometry.GetGeometryType() !=
        geometry::Geometry::GeometryType::TriangleMesh) {
        PrintShaderWarning("Rendering type is not geometry::TriangleMesh.");
        return false;
    }
    const geometry::TriangleMesh &mesh =
            (const geometry::TriangleMesh &)geometry;
    if (mesh.HasTriangles() == false) {
        PrintShaderWarning("Binding failed with empty triangle mesh.");
        return false;
    }
    if (mesh.HasTriangleNormals() == false ||
        mesh.HasVertexNormals() == false) {
        PrintShaderWarning("Binding failed because mesh has no normals.");
        PrintShaderWarning("Call ComputeVertexNormals() before binding.");
        return false;
    }
    copy_trianglemesh_functor func(
            thrust::raw_pointer_cast(mesh.vertices_.data()),
            thrust::raw_pointer_cast(mesh.vertex_normals_.data()),
            (int *)(thrust::raw_pointer_cast(mesh.triangles_.data())),
            thrust::raw_pointer_cast(mesh.triangle_normals_.data()),
            option.mesh_shade_option_);
    thrust::transform(
            thrust::make_counting_iterator<size_t>(0),
            thrust::make_counting_iterator(mesh.triangles_.size() * 3),
            make_tuple_iterator(points, normals), func);
    draw_arrays_mode_ = GL_TRIANGLES;
    draw_arrays_size_ = GLsizei(mesh.triangles_.size() * 3);
    return true;
}

size_t NormalShaderForTriangleMesh::GetDataSize(
        const geometry::Geometry &geometry) const {
    return ((const geometry::TriangleMesh &)geometry).triangles_.size() * 3;
}