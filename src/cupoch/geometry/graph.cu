#include "hip/hip_runtime.h"
#include "cupoch/geometry/graph.h"
#include "cupoch/geometry/geometry_functor.h"

#include <thrust/gather.h>
#include <thrust/iterator/discard_iterator.h>

namespace cupoch {
namespace geometry {

namespace {

struct replace_color_functor {
    replace_color_functor(const Eigen::Vector2i* lines,
                          Eigen::Vector3f* colors,
                          const Eigen::Vector2i& edge,
                          const Eigen::Vector3f& color,
                          bool is_directed)
                          : lines_(lines), colors_(colors),
                          edge_(edge), color_(color),
                          is_directed_(is_directed) {};
    const Eigen::Vector2i* lines_;
    Eigen::Vector3f* colors_;
    const Eigen::Vector2i edge_;
    const Eigen::Vector3f color_;
    const bool is_directed_;
    __device__ void operator() (size_t idx) const {
        if (lines_[idx] == edge_ || (!is_directed_ && lines_[idx] == Eigen::Vector2i(edge_[1], edge_[0]))) {
            colors_[idx] = color_;
        }
    }
};

struct replace_colors_functor {
    replace_colors_functor(Eigen::Vector3f* colors,
                           const Eigen::Vector3f& color)
                           : colors_(colors),
                           color_(color) {};
    Eigen::Vector3f* colors_;
    const Eigen::Vector3f color_;
    __device__ void operator() (size_t idx) const {
        colors_[idx] = color_;
    }
};

struct relax_functor {
    relax_functor(const Eigen::Vector2i* lines,
                  const int* edge_index_offsets,
                  const float* edge_weights,
                  int* open_flags,
                  const Graph::SSSPResult* res,
                  Graph::SSSPResult* res_tmp)
                  : lines_(lines), edge_index_offsets_(edge_index_offsets),
                  edge_weights_(edge_weights), open_flags_(open_flags),
                  res_(res), res_tmp_(res_tmp) {};
    const Eigen::Vector2i* lines_;
    const int* edge_index_offsets_;
    const float* edge_weights_;
    int* open_flags_;
    const Graph::SSSPResult* res_;
    Graph::SSSPResult* res_tmp_;
    __device__ void operator() (size_t idx) {
        if (open_flags_[idx] == 0) return;
        open_flags_[idx] = 0;
        int s_edge = edge_index_offsets_[idx];
        int e_edge = edge_index_offsets_[idx + 1];
        for (int j = s_edge; j < e_edge; ++j) {
            int k = lines_[j][0];
            res_tmp_[j].shortest_distance_ = res_[k].shortest_distance_ + edge_weights_[j];
            res_tmp_[j].prev_index_ = k;
        }
    }
};

struct update_shortest_distances_functor {
    update_shortest_distances_functor(int* open_flags,
                                      Graph::SSSPResult* res,
                                      const Graph::SSSPResult* res_tmp)
                                      : open_flags_(open_flags),
                                      res_(res), res_tmp_(res_tmp) {};
    int* open_flags_;
    Graph::SSSPResult* res_;
    const Graph::SSSPResult* res_tmp_;
    __device__ void operator() (size_t idx) {
        if (res_[idx].shortest_distance_ > res_tmp_[idx].shortest_distance_) {
            res_[idx] = res_tmp_[idx];
            open_flags_[idx] = 1;
        }
    }
};

}

Graph::Graph() : LineSet(Geometry::GeometryType::Graph) {}
Graph::Graph(const utility::device_vector<Eigen::Vector3f> &points)
 : LineSet(Geometry::GeometryType::Graph, points, utility::device_vector<Eigen::Vector2i>()) {}
Graph::Graph(const thrust::host_vector<Eigen::Vector3f> &points)
 : LineSet(Geometry::GeometryType::Graph, points, utility::device_vector<Eigen::Vector2i>()) { ConstructGraph(); }
Graph::~Graph() {}
 Graph::Graph(const Graph &other)
 : LineSet(Geometry::GeometryType::Graph, other.points_, other.lines_),
 edge_index_offsets_(other.edge_index_offsets_), edge_weights_(other.edge_weights_),
 is_directed_(other.is_directed_) {}

thrust::host_vector<int> Graph::GetEdgeIndexOffsets() const {
    thrust::host_vector<int> edge_index_offsets = edge_index_offsets_;
    return edge_index_offsets;
}

void Graph::SetEdgeIndexOffsets(const thrust::host_vector<int>& edge_index_offsets) {
    edge_index_offsets_ = edge_index_offsets;
}

thrust::host_vector<float> Graph::GetEdgeWeights() const {
    thrust::host_vector<float> edge_weights = edge_weights_;
    return edge_weights;
}

void Graph::SetEdgeWeights(const thrust::host_vector<float>& edge_weights) {
    edge_weights_ = edge_weights;
}

Graph &Graph::Clear() {
    LineSet::Clear();
    edge_index_offsets_.clear();
    edge_weights_.clear();
    return *this;
}

Graph &Graph::ConstructGraph() {
    if (lines_.empty()) {
        utility::LogError("[ConstructGraph] Graph has no edges.");
        return *this;
    }

    bool has_colors = HasColors();
    bool has_weights = HasWeights();
    if (has_colors && has_weights) {
        thrust::sort_by_key(lines_.begin(), lines_.end(),
                            make_tuple_iterator(edge_weights_.begin(), colors_.begin()));
    } else if (!has_colors && has_weights) {
        thrust::sort_by_key(lines_.begin(), lines_.end(), edge_weights_.begin());
    } else if (has_colors && !has_weights) {
        thrust::sort_by_key(lines_.begin(), lines_.end(), colors_.begin());
    } else {
        thrust::sort(lines_.begin(), lines_.end());
        edge_weights_.resize(lines_.size(), 1.0);
    }
    edge_index_offsets_.resize(points_.size() + 1, 0);
    utility::device_vector<int> indices(lines_.size());
    utility::device_vector<int> counts(lines_.size());
    const auto begin = thrust::make_transform_iterator(lines_.begin(), extract_element_functor<int, 2, 0>());
    auto end = thrust::reduce_by_key(begin, begin + lines_.size(), thrust::make_constant_iterator<int>(1),
                                     indices.begin(), counts.begin());
    indices.resize(thrust::distance(indices.begin(), end.first));
    counts.resize(thrust::distance(counts.begin(), end.second));
    thrust::gather(indices.begin(), indices.end(), counts.begin(), edge_index_offsets_.begin());
    thrust::exclusive_scan(edge_index_offsets_.begin(), edge_index_offsets_.end(), edge_index_offsets_.begin());
    return *this;
}

Graph &Graph::AddEdge(const Eigen::Vector2i &edge, float weight) {
    lines_.push_back(edge);
    edge_weights_.push_back(weight);
    if (!is_directed_) {
        lines_.push_back(Eigen::Vector2i(edge[1], edge[0]));
        edge_weights_.push_back(weight);
    }
    if (HasColors()) {
        colors_.push_back(Eigen::Vector3f::Ones());
        if (!is_directed_) colors_.push_back(Eigen::Vector3f::Ones());
    }
    return ConstructGraph();
}

Graph &Graph::AddEdges(const utility::device_vector<Eigen::Vector2i> &edges,
                       const utility::device_vector<float> &weights) {
    if (!weights.empty() && edges.size() != weights.size()) {
        utility::LogError("[AddEdges] edges size is not equal to weights size.");
        return *this;
    }
    size_t n_old_lines = lines_.size();
    lines_.insert(lines_.end(), edges.begin(), edges.end());
    if (!is_directed_) {
        lines_.insert(lines_.end(), thrust::make_transform_iterator(edges.begin(), reverse_index_functor<int>()),
                      thrust::make_transform_iterator(edges.end(), reverse_index_functor<int>()));
    }
    if (weights.empty()) {
        if (!is_directed_) {
            edge_weights_.resize(2 * lines_.size());
        } else {
            edge_weights_.resize(lines_.size());
        }
        thrust::fill(edge_weights_.begin() + n_old_lines, edge_weights_.end(), 1.0);
    } else {
        edge_weights_.insert(edge_weights_.end(), weights.begin(), weights.end());
        if (!is_directed_) edge_weights_.insert(edge_weights_.end(), weights.begin(), weights.end());
    }
    if (HasColors()) {
        colors_.resize(lines_.size());
        thrust::fill(colors_.begin() + n_old_lines, colors_.end(), Eigen::Vector3f::Ones());
    }
    return ConstructGraph();
}

Graph &Graph::AddEdges(const thrust::host_vector<Eigen::Vector2i> &edges,
                       const thrust::host_vector<float> &weights) {
    utility::device_vector<Eigen::Vector2i> d_edges = edges;
    utility::device_vector<float> d_weights = weights;
    return AddEdges(d_edges, d_weights);
}

Graph &Graph::RemoveEdge(const Eigen::Vector2i &edge) {
    bool has_colors = HasColors();
    bool has_weights = HasWeights();
    if (has_colors && has_weights) {
        auto begin = make_tuple_iterator(lines_.begin(), edge_weights_.begin(), colors_.begin());
        auto end = thrust::remove_if(begin,
                make_tuple_iterator(lines_.end(), edge_weights_.end(), colors_.end()),
                [edge, is_directed = is_directed_] __device__ (const thrust::tuple<Eigen::Vector2i, float, Eigen::Vector3f> &x) {
                    const Eigen::Vector2i& l = thrust::get<0>(x);
                    return l == edge || (!is_directed && l == Eigen::Vector2i(edge[1], edge[0]));
                });
        size_t n_out = thrust::distance(begin, end);
        lines_.resize(n_out);
        edge_weights_.resize(n_out);
        colors_.resize(n_out);
    } else if (has_colors && !has_weights) {
        auto begin = make_tuple_iterator(lines_.begin(), colors_.begin());
        auto end = thrust::remove_if(begin,
                make_tuple_iterator(lines_.end(), colors_.end()),
                [edge, is_directed = is_directed_] __device__ (const thrust::tuple<Eigen::Vector2i, Eigen::Vector3f> &x) {
                    const Eigen::Vector2i& l = thrust::get<0>(x);
                    return l == edge || (!is_directed && l == Eigen::Vector2i(edge[1], edge[0]));
                });
        size_t n_out = thrust::distance(begin, end);
        lines_.resize(n_out);
        colors_.resize(n_out);
    } else if (!has_colors && has_weights) {
        auto begin = make_tuple_iterator(lines_.begin(), edge_weights_.begin());
        auto end = thrust::remove_if(begin,
                make_tuple_iterator(lines_.end(), edge_weights_.end()),
                [edge, is_directed = is_directed_] __device__ (const thrust::tuple<Eigen::Vector2i, float> &x) {
                    const Eigen::Vector2i& l = thrust::get<0>(x);
                    return l == edge || (!is_directed && l == Eigen::Vector2i(edge[1], edge[0]));
                });
        size_t n_out = thrust::distance(begin, end);
        lines_.resize(n_out);
        edge_weights_.resize(n_out);
    } else {
        auto end = thrust::remove_if(lines_.begin(), lines_.end(),
            [edge, is_directed = is_directed_] __device__ (const Eigen::Vector2i &l) {
                return l == edge || (!is_directed && l == Eigen::Vector2i(edge[1], edge[0]));
            });
        lines_.resize(thrust::distance(lines_.begin(), end));
    }
    return ConstructGraph();
}

Graph &Graph::RemoveEdges(const utility::device_vector<Eigen::Vector2i> &edges) {
    bool has_colors = HasColors();
    bool has_weights = HasWeights();
    utility::device_vector<Eigen::Vector2i> new_lines;
    utility::device_vector<float> new_weights;
    utility::device_vector<Eigen::Vector3f> new_colors;
    utility::device_vector<Eigen::Vector2i> sorted_edges = edges;
    thrust::sort(sorted_edges.begin(), sorted_edges.end());
    auto cnst_w = thrust::make_constant_iterator<float>(1.0);
    auto cnst_c = thrust::make_constant_iterator<Eigen::Vector3f>(Eigen::Vector3f::Ones());
    if (has_colors && has_weights) {
        auto func = tuple_element_compare_functor<thrust::tuple<Eigen::Vector2i, float, Eigen::Vector3f>, 0, thrust::greater<Eigen::Vector2i>>();
        auto begin = make_tuple_iterator(new_lines.begin(), new_weights.end(), new_colors.begin());
        auto end1 = thrust::set_difference(make_tuple_iterator(lines_.begin(), edge_weights_.begin(), colors_.begin()),
                make_tuple_iterator(lines_.end(), edge_weights_.end(), colors_.end()),
                make_tuple_iterator(sorted_edges.begin(), cnst_w, cnst_c),
                make_tuple_iterator(sorted_edges.end(), cnst_w, cnst_c),
                begin, func);
        size_t n_out1 = thrust::distance(begin, end1);
        new_lines.resize(n_out1);
        new_weights.resize(n_out1);
        new_colors.resize(n_out1);
        if (!is_directed_) {
            auto end2 = thrust::set_difference(make_tuple_iterator(lines_.begin(), edge_weights_.begin(), colors_.begin()),
                    make_tuple_iterator(lines_.end(), edge_weights_.end(), colors_.end()),
                    make_tuple_iterator(thrust::make_transform_iterator(sorted_edges.begin(), reverse_index_functor<int>()), cnst_w, cnst_c),
                    make_tuple_iterator(thrust::make_transform_iterator(sorted_edges.end(), reverse_index_functor<int>()), cnst_w, cnst_c),
                    begin, func);
            size_t n_out2 = thrust::distance(begin, end1);
            new_lines.resize(n_out2);
            new_weights.resize(n_out2);
            new_colors.resize(n_out2);
        }
    } else if (has_colors && !has_weights) {
        auto func = tuple_element_compare_functor<thrust::tuple<Eigen::Vector2i, Eigen::Vector3f>, 0, thrust::greater<Eigen::Vector2i>>();
        auto begin = make_tuple_iterator(new_lines.begin(), new_colors.end());
        auto end1 = thrust::set_difference(make_tuple_iterator(lines_.begin(), colors_.begin()),
                make_tuple_iterator(lines_.end(), colors_.end()),
                make_tuple_iterator(sorted_edges.begin(), cnst_c),
                make_tuple_iterator(sorted_edges.end(), cnst_c),
                begin, func);
        size_t n_out1 = thrust::distance(begin, end1);
        new_lines.resize(n_out1);
        new_colors.resize(n_out1);
        if (!is_directed_) {
            auto end2 = thrust::set_difference(make_tuple_iterator(lines_.begin(), colors_.begin()),
                    make_tuple_iterator(lines_.end(), colors_.end()),
                    make_tuple_iterator(thrust::make_transform_iterator(sorted_edges.begin(), reverse_index_functor<int>()), cnst_c),
                    make_tuple_iterator(thrust::make_transform_iterator(sorted_edges.end(), reverse_index_functor<int>()), cnst_c),
                    begin, func);
            size_t n_out2 = thrust::distance(begin, end1);
            new_lines.resize(n_out2);
            new_colors.resize(n_out2);
        }
    } else if (!has_colors && has_weights) {
        auto func = tuple_element_compare_functor<thrust::tuple<Eigen::Vector2i, float>, 0, thrust::greater<Eigen::Vector2i>>();
        auto begin = make_tuple_iterator(new_lines.begin(), new_weights.end());
        auto end1 = thrust::set_difference(make_tuple_iterator(lines_.begin(), edge_weights_.begin()),
                make_tuple_iterator(lines_.end(), edge_weights_.end()),
                make_tuple_iterator(sorted_edges.begin(), cnst_w),
                make_tuple_iterator(sorted_edges.end(), cnst_w),
                begin, func);
        size_t n_out1 = thrust::distance(begin, end1);
        new_lines.resize(n_out1);
        new_weights.resize(n_out1);
        if (!is_directed_) {
            auto end2 = thrust::set_difference(make_tuple_iterator(lines_.begin(), edge_weights_.begin()),
                    make_tuple_iterator(lines_.end(), edge_weights_.end()),
                    make_tuple_iterator(thrust::make_transform_iterator(sorted_edges.begin(), reverse_index_functor<int>()), cnst_w),
                    make_tuple_iterator(thrust::make_transform_iterator(sorted_edges.end(), reverse_index_functor<int>()), cnst_w),
                    begin, func);
            size_t n_out2 = thrust::distance(begin, end1);
            new_lines.resize(n_out2);
            new_weights.resize(n_out2);
        }
    } else {
        auto end1 = thrust::set_difference(lines_.begin(), lines_.end(),
                sorted_edges.begin(), sorted_edges.end(), new_lines.begin());
        new_lines.resize(thrust::distance(new_lines.begin(), end1));
        if (!is_directed_) {
            auto end2 = thrust::set_difference(lines_.begin(), lines_.end(),
                    thrust::make_transform_iterator(sorted_edges.begin(), reverse_index_functor<int>()),
                    thrust::make_transform_iterator(sorted_edges.end(), reverse_index_functor<int>()),
                    new_lines.begin());
            new_lines.resize(thrust::distance(new_lines.begin(), end2));
        }
    }
    lines_ = new_lines;
    edge_weights_ = new_weights;
    colors_ = new_colors;
    return ConstructGraph();
}

Graph &Graph::RemoveEdges(const thrust::host_vector<Eigen::Vector2i> &edges) {
    utility::device_vector<Eigen::Vector2i> d_edges = edges;
    return RemoveEdges(d_edges);
}

Graph &Graph::PaintEdgeColor(const Eigen::Vector2i &edge, const Eigen::Vector3f &color) {
    if (!HasColors()) {
        colors_.resize(lines_.size(), Eigen::Vector3f::Ones());
    }
    replace_color_functor func(thrust::raw_pointer_cast(lines_.data()),
                               thrust::raw_pointer_cast(colors_.data()),
                               edge, color, is_directed_);
    thrust::for_each(thrust::make_counting_iterator<size_t>(0), thrust::make_counting_iterator(lines_.size()), func);
    return *this;
}

Graph &Graph::PaintEdgesColor(const utility::device_vector<Eigen::Vector2i> &edges, const Eigen::Vector3f &color) {
    utility::device_vector<Eigen::Vector2i> sorted_edges = edges;
    utility::device_vector<size_t> indices(edges.size());
    thrust::sort(sorted_edges.begin(), sorted_edges.end());
    thrust::set_intersection(make_tuple_iterator(lines_.begin(), thrust::make_counting_iterator<size_t>(0)),
            make_tuple_iterator(lines_.end(), thrust::make_counting_iterator(lines_.size())),
            make_tuple_iterator(sorted_edges.begin(), thrust::make_constant_iterator<size_t>(0)),
            make_tuple_iterator(sorted_edges.end(), thrust::make_constant_iterator<size_t>(0)),
            make_tuple_iterator(thrust::make_discard_iterator(), indices.begin()),
            tuple_element_compare_functor<thrust::tuple<Eigen::Vector2i, size_t>, 0, thrust::greater<Eigen::Vector2i>>());
    replace_colors_functor func(thrust::raw_pointer_cast(colors_.data()), color);
    thrust::for_each(indices.begin(), indices.end(), func);
    if (!is_directed_) {
        thrust::transform(sorted_edges.begin(), sorted_edges.end(), sorted_edges.begin(), reverse_index_functor<int>());
        thrust::sort(sorted_edges.begin(), sorted_edges.end());
        thrust::set_intersection(make_tuple_iterator(lines_.begin(), thrust::make_counting_iterator<size_t>(0)),
                make_tuple_iterator(lines_.end(), thrust::make_counting_iterator(lines_.size())),
                make_tuple_iterator(sorted_edges.begin(), thrust::make_constant_iterator<size_t>(0)),
                make_tuple_iterator(sorted_edges.end(), thrust::make_constant_iterator<size_t>(0)),
                make_tuple_iterator(thrust::make_discard_iterator(), indices.begin()),
                tuple_element_compare_functor<thrust::tuple<Eigen::Vector2i, size_t>, 0, thrust::greater<Eigen::Vector2i>>());
        thrust::for_each(indices.begin(), indices.end(), func);
    }
    return *this;
}

Graph &Graph::PaintEdgesColor(const thrust::host_vector<Eigen::Vector2i> &edges, const Eigen::Vector3f &color) {
    utility::device_vector<Eigen::Vector2i> d_edges = edges;
    return PaintEdgesColor(d_edges, color);
}

Graph &Graph::SetEdgeWeightsFromDistance() {
    edge_weights_.resize(lines_.size());
    Eigen::Vector3f *pt_ptr = thrust::raw_pointer_cast(points_.data());
    thrust::transform(lines_.begin(), lines_.end(), edge_weights_.begin(),
                      [pt_ptr] __device__ (const Eigen::Vector2i& edge) {
                          return (pt_ptr[edge[0]] - pt_ptr[edge[1]]).norm();
                      });
    return *this;
}

Graph::SSSPResultArray Graph::DijkstraPath(int start_node_index) const {
    SSSPResultArray out(points_.size());

    if (!IsConstructed()) {
        utility::LogError("[DijkstraPath] this graph is not constructed.");
        return out;
    }

    utility::device_vector<int> open_flags(points_.size(), 0);
    utility::device_vector<int> indices(lines_.size());
    utility::device_vector<Eigen::Vector2i> sorted_lines(lines_.size());
    SSSPResultArray res_tmp_line(lines_.size());
    SSSPResultArray sorted_res_tmp_line(lines_.size());
    SSSPResultArray sorted_res_tmp_line_s(lines_.size());
    SSSPResultArray res_tmp(points_.size());
    open_flags[start_node_index] = 1;
    out[start_node_index] = SSSPResult(0.0);
    relax_functor func1(thrust::raw_pointer_cast(lines_.data()),
                        thrust::raw_pointer_cast(edge_index_offsets_.data()),
                        thrust::raw_pointer_cast(edge_weights_.data()),
                        thrust::raw_pointer_cast(open_flags.data()),
                        thrust::raw_pointer_cast(out.data()),
                        thrust::raw_pointer_cast(res_tmp_line.data()));
    update_shortest_distances_functor func2(thrust::raw_pointer_cast(open_flags.data()),
                                            thrust::raw_pointer_cast(out.data()),
                                            thrust::raw_pointer_cast(res_tmp.data()));
    while (thrust::reduce(open_flags.begin(), open_flags.end()) > 0) {
        thrust::for_each(thrust::make_counting_iterator<size_t>(0),
                         thrust::make_counting_iterator(points_.size()), func1);
        sorted_lines = lines_;
        sorted_res_tmp_line = res_tmp_line;
        thrust::sort_by_key(sorted_lines.begin(), sorted_lines.end(), sorted_res_tmp_line.begin(),
                            [] __device__ (const Eigen::Vector2i &lhs, const Eigen::Vector2i &rhs) {
                                return lhs[1] < rhs[1];
                            });
        const auto begin = thrust::make_transform_iterator(sorted_lines.begin(), extract_element_functor<int, 2, 1>());
        auto end = thrust::reduce_by_key(begin, begin + sorted_lines.size(), sorted_res_tmp_line.begin(),
                                         indices.begin(), sorted_res_tmp_line_s.begin(),
                                         thrust::equal_to<int>(),
                                         [] __device__ (const SSSPResult& lhs, const SSSPResult& rhs) {
                                             return (lhs.shortest_distance_<= rhs.shortest_distance_) ? lhs : rhs;
                                         });
        size_t ns = thrust::distance(indices.begin(), end.first);
        thrust::gather(indices.begin(), indices.begin() + ns, sorted_res_tmp_line_s.begin(), res_tmp.begin());
        thrust::for_each(thrust::make_counting_iterator<size_t>(0),
                         thrust::make_counting_iterator(points_.size()), func2);
    }
    return out;
}

Graph::SSSPResultHostArray Graph::DijkstraPathHost(int start_node_index) const {
    auto out = DijkstraPath(start_node_index);
    SSSPResultHostArray h_out = out;
    return h_out;
}

thrust::host_vector<int> Graph::DijkstraPath(int start_node_index, int end_node_index) const {
    auto res = DijkstraPath(start_node_index);
    SSSPResultHostArray h_res = res;
    if (h_res[end_node_index].prev_index_ < 0) return thrust::host_vector<int>();
    thrust::host_vector<int> path_nodes;
    path_nodes.push_back(end_node_index);
    int prev_index = h_res[end_node_index].prev_index_;
    while (prev_index != start_node_index) {
        path_nodes.push_back(prev_index);
        prev_index = h_res[prev_index].prev_index_;
    }
    path_nodes.push_back(start_node_index);
    thrust::reverse(path_nodes.begin(), path_nodes.end());
    return path_nodes;
}

}
}