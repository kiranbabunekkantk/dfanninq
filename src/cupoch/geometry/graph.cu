#include "hip/hip_runtime.h"
#include "cupoch/geometry/graph.h"
#include "cupoch/geometry/geometry_functor.h"
#include "cupoch/geometry/kdtree_flann.h"
#include "cupoch/utility/console.h"

#include <thrust/gather.h>
#include <thrust/iterator/discard_iterator.h>

namespace cupoch {
namespace geometry {

namespace {

struct extract_near_edges_functor {
    extract_near_edges_functor(const Eigen::Vector3f &point,
                               int point_no,
                               float max_edge_distance)
                               : point_(point),
                               point_no_(point_no),
                               max_edge_distance_(max_edge_distance) {};
    const Eigen::Vector3f point_;
    const int point_no_;
    const float max_edge_distance_;
    __device__ thrust::tuple<Eigen::Vector2i, float> operator() (const thrust::tuple<int, Eigen::Vector3f>& x) const {
        int i = thrust::get<0>(x);
        const Eigen::Vector3f& p = thrust::get<1>(x);
        float d = (p - point_).norm();
        return thrust::make_tuple((d < max_edge_distance_) ? Eigen::Vector2i(i, point_no_) : Eigen::Vector2i(-1, -1), d);
    }
};

struct relax_functor {
    relax_functor(const Eigen::Vector2i* lines,
                  const int* edge_index_offsets,
                  const float* edge_weights,
                  const int* edge_table,
                  int* open_flags,
                  const Graph::SSSPResult* res,
                  Graph::SSSPResult* res_tmp)
                  : lines_(lines), edge_index_offsets_(edge_index_offsets),
                  edge_weights_(edge_weights), edge_table_(edge_table),
                  open_flags_(open_flags), res_(res), res_tmp_(res_tmp) {};
    const Eigen::Vector2i* lines_;
    const int* edge_index_offsets_;
    const float* edge_weights_;
    const int* edge_table_;
    int* open_flags_;
    const Graph::SSSPResult* res_;
    Graph::SSSPResult* res_tmp_;
    __device__ void operator() (size_t idx) {
        if (open_flags_[idx] == 0) return;
        open_flags_[idx] = 0;
        int s_edge = edge_index_offsets_[idx];
        int e_edge = edge_index_offsets_[idx + 1];
        for (int j = s_edge; j < e_edge; ++j) {
            int k = lines_[j][0];
            res_tmp_[edge_table_[j]].shortest_distance_ = res_[k].shortest_distance_ + edge_weights_[j];
            res_tmp_[edge_table_[j]].prev_index_ = k;
        }
    }
};

struct update_shortest_distances_functor {
    update_shortest_distances_functor(int* open_flags,
                                      Graph::SSSPResult* res,
                                      const Graph::SSSPResult* res_tmp)
                                      : open_flags_(open_flags),
                                      res_(res), res_tmp_(res_tmp) {};
    int* open_flags_;
    Graph::SSSPResult* res_;
    const Graph::SSSPResult* res_tmp_;
    __device__ void operator() (size_t idx) {
        if (res_[idx].shortest_distance_ > res_tmp_[idx].shortest_distance_) {
            res_[idx] = res_tmp_[idx];
            open_flags_[idx] = 1;
        }
    }
};

struct compare_path_length_functor {
    compare_path_length_functor(const Graph::SSSPResult* res,
                                const int* open_flags, int end_node_index)
    : res_(res), open_flags_(open_flags), end_node_index_(end_node_index) {}
    const Graph::SSSPResult* res_;
    const int* open_flags_;
    const int end_node_index_;
    __device__ bool operator()(size_t idx) const {
        return (open_flags_[idx] && res_[idx].shortest_distance_ < res_[end_node_index_].shortest_distance_);
    }
};


template <class... Args>
struct check_edge_functor {
    check_edge_functor(const Eigen::Vector2i& edge, bool is_directed)
    : edge_(edge), is_directed_(is_directed) {};
    const Eigen::Vector2i edge_;
    const bool is_directed_;
    __device__ bool operator() (const thrust::tuple<Args...> &x) const {
        const Eigen::Vector2i& l = thrust::get<0>(x);
        return l == edge_ || (!is_directed_ && l == Eigen::Vector2i(edge_[1], edge_[0]));
    }
};

}

Graph::Graph() : LineSet(Geometry::GeometryType::Graph) {}
Graph::Graph(const utility::device_vector<Eigen::Vector3f> &points)
 : LineSet(Geometry::GeometryType::Graph, points, utility::device_vector<Eigen::Vector2i>()) {}
Graph::Graph(const thrust::host_vector<Eigen::Vector3f> &points)
 : LineSet(Geometry::GeometryType::Graph, points, utility::device_vector<Eigen::Vector2i>()) { ConstructGraph(); }
Graph::~Graph() {}
 Graph::Graph(const Graph &other)
 : LineSet(Geometry::GeometryType::Graph, other.points_, other.lines_),
 edge_index_offsets_(other.edge_index_offsets_), edge_weights_(other.edge_weights_),
 is_directed_(other.is_directed_) {}

thrust::host_vector<int> Graph::GetEdgeIndexOffsets() const {
    thrust::host_vector<int> edge_index_offsets = edge_index_offsets_;
    return edge_index_offsets;
}

void Graph::SetEdgeIndexOffsets(const thrust::host_vector<int>& edge_index_offsets) {
    edge_index_offsets_ = edge_index_offsets;
}

thrust::host_vector<float> Graph::GetEdgeWeights() const {
    thrust::host_vector<float> edge_weights = edge_weights_;
    return edge_weights;
}

void Graph::SetEdgeWeights(const thrust::host_vector<float>& edge_weights) {
    edge_weights_ = edge_weights;
}

Graph &Graph::Clear() {
    LineSet::Clear();
    edge_index_offsets_.clear();
    edge_weights_.clear();
    return *this;
}

Graph &Graph::ConstructGraph(bool set_edge_weights_from_distance) {
    if (lines_.empty()) {
        utility::LogError("[ConstructGraph] Graph has no edges.");
        return *this;
    }

    bool has_colors = HasColors();
    bool has_weights = HasWeights();
    if (has_colors && has_weights) {
        thrust::sort_by_key(lines_.begin(), lines_.end(),
                            make_tuple_begin(edge_weights_, colors_));
    } else if (!has_colors && has_weights) {
        thrust::sort_by_key(lines_.begin(), lines_.end(), edge_weights_.begin());
    } else if (has_colors && !has_weights) {
        thrust::sort_by_key(lines_.begin(), lines_.end(), colors_.begin());
    } else {
        thrust::sort(lines_.begin(), lines_.end());
        edge_weights_.resize(lines_.size(), 1.0);
    }
    edge_index_offsets_.resize(points_.size() + 1, 0);
    utility::device_vector<int> indices(lines_.size());
    utility::device_vector<int> counts(lines_.size());
    const auto begin = thrust::make_transform_iterator(lines_.begin(), extract_element_functor<int, 2, 0>());
    auto end = thrust::reduce_by_key(begin, begin + lines_.size(), thrust::make_constant_iterator<int>(1),
                                     indices.begin(), counts.begin());
    indices.resize(thrust::distance(indices.begin(), end.first));
    counts.resize(thrust::distance(counts.begin(), end.second));
    thrust::gather(indices.begin(), indices.end(), counts.begin(), edge_index_offsets_.begin());
    thrust::exclusive_scan(edge_index_offsets_.begin(), edge_index_offsets_.end(), edge_index_offsets_.begin());
    if (set_edge_weights_from_distance) {
        SetEdgeWeightsFromDistance();
    }
    return *this;
}

Graph &Graph::ConnectToNearestNeighbors(float max_edge_distance, int max_num_edges) {
    utility::device_vector<int> indices;
    utility::device_vector<float> weights;
    utility::device_vector<Eigen::Vector2i> new_edges(points_.size() * (max_num_edges + 1));
    geometry::KDTreeFlann kdtree;
    kdtree.SetRawData(points_);
    kdtree.SearchHybrid(points_, max_edge_distance, max_num_edges + 1, indices, weights);
    thrust::transform(thrust::make_counting_iterator(0),
                      thrust::make_counting_iterator<int>(new_edges.size()),
                      indices.begin(), new_edges.begin(),
                      [max_num_edges] __device__ (int idx, int j) {
                          int i = idx / max_num_edges;
                          return (j >= 0 && i != j) ? Eigen::Vector2i(i, j) : Eigen::Vector2i(-1, -1);
                      });
    auto remove_fn = [] __device__ (const thrust::tuple<Eigen::Vector2i, float>& x) {
        return thrust::get<0>(x)[0] < 0;
    };
    remove_if_vectors(remove_fn, new_edges, weights);
    thrust::sort_by_key(new_edges.begin(), new_edges.end(), weights.begin());
    utility::device_vector<Eigen::Vector2i> res_edges(new_edges.size());
    utility::device_vector<float> res_weights(new_edges.size());
    auto func = tuple_element_compare_functor<EdgeWeight, 0, thrust::greater<Eigen::Vector2i>>();
    auto begin = make_tuple_begin(res_edges, res_weights);
    auto end = thrust::set_difference(make_tuple_begin(new_edges, weights),
                                      make_tuple_end(new_edges, weights),
                                      make_tuple_begin(lines_, edge_weights_),
                                      make_tuple_end(lines_, edge_weights_),
                                      begin, func);
    resize_all(thrust::distance(begin, end), res_edges, res_weights);
    lines_.insert(lines_.end(), res_edges.begin(), res_edges.end());
    edge_weights_.insert(edge_weights_.end(), res_weights.begin(), res_weights.end());
    return ConstructGraph(false);
}

Graph &Graph::AddNodeAndConnect(const Eigen::Vector3f& point, float max_edge_distance, bool lazy_add) {
    size_t n_points = points_.size();
    utility::device_vector<Eigen::Vector2i> new_edges(n_points);
    utility::device_vector<float> new_weights(n_points);
    extract_near_edges_functor func(point, n_points, max_edge_distance);
    thrust::transform(make_tuple_iterator(thrust::make_counting_iterator(0), points_.begin()),
                      make_tuple_iterator(thrust::make_counting_iterator<int>(n_points), points_.end()),
                      make_tuple_begin(new_edges, new_weights), func);
    auto remove_fn = [] __device__ (const thrust::tuple<Eigen::Vector2i, float>& x) {
        return thrust::get<0>(x)[0] < 0;
    };
    remove_if_vectors(remove_fn, new_edges, new_weights);
    points_.push_back(point);
    return AddEdges(new_edges, new_weights, lazy_add);
}

Graph &Graph::AddEdge(const Eigen::Vector2i &edge, float weight, bool lazy_add) {
    lines_.push_back(edge);
    edge_weights_.push_back(weight);
    if (!is_directed_) {
        lines_.push_back(Eigen::Vector2i(edge[1], edge[0]));
        edge_weights_.push_back(weight);
    }
    if (HasColors()) {
        colors_.push_back(Eigen::Vector3f::Ones());
        if (!is_directed_) colors_.push_back(Eigen::Vector3f::Ones());
    }
    return (lazy_add) ? *this : ConstructGraph(false);
}

Graph &Graph::AddEdges(const utility::device_vector<Eigen::Vector2i> &edges,
                       const utility::device_vector<float> &weights, bool lazy_add) {
    if (!weights.empty() && edges.size() != weights.size()) {
        utility::LogError("[AddEdges] edges size is not equal to weights size.");
        return *this;
    }
    size_t n_old_lines = lines_.size();
    lines_.insert(lines_.end(), edges.begin(), edges.end());
    if (!is_directed_) {
        lines_.insert(lines_.end(), thrust::make_transform_iterator(edges.begin(), swap_index_functor<int>()),
                      thrust::make_transform_iterator(edges.end(), swap_index_functor<int>()));
    }
    if (weights.empty()) {
        if (!is_directed_) {
            edge_weights_.resize(2 * lines_.size());
        } else {
            edge_weights_.resize(lines_.size());
        }
        thrust::fill(edge_weights_.begin() + n_old_lines, edge_weights_.end(), 1.0);
    } else {
        edge_weights_.insert(edge_weights_.end(), weights.begin(), weights.end());
        if (!is_directed_) edge_weights_.insert(edge_weights_.end(), weights.begin(), weights.end());
    }
    if (HasColors()) {
        colors_.resize(lines_.size());
        thrust::fill(colors_.begin() + n_old_lines, colors_.end(), Eigen::Vector3f::Ones());
    }
    return (lazy_add) ? *this : ConstructGraph(false);
}

Graph &Graph::AddEdges(const thrust::host_vector<Eigen::Vector2i> &edges,
                       const thrust::host_vector<float> &weights,
                       bool lazy_add) {
    utility::device_vector<Eigen::Vector2i> d_edges = edges;
    utility::device_vector<float> d_weights = weights;
    return AddEdges(d_edges, d_weights, lazy_add);
}

Graph &Graph::RemoveEdge(const Eigen::Vector2i &edge) {
    bool has_colors = HasColors();
    bool has_weights = HasWeights();
    if (has_colors && has_weights) {
        remove_if_vectors(check_edge_functor<Eigen::Vector2i, float, Eigen::Vector3f>(edge, is_directed_),
                lines_, edge_weights_, colors_);
    } else if (has_colors && !has_weights) {
        remove_if_vectors(check_edge_functor<Eigen::Vector2i, Eigen::Vector3f>(edge, is_directed_), lines_, colors_);
    } else if (!has_colors && has_weights) {
        remove_if_vectors(check_edge_functor<Eigen::Vector2i, float>(edge, is_directed_), lines_, edge_weights_);
    } else {
        remove_if_vectors(check_edge_functor<Eigen::Vector2i>(edge, is_directed_), lines_);
    }
    return ConstructGraph(false);
}

Graph &Graph::RemoveEdges(const utility::device_vector<Eigen::Vector2i> &edges) {
    bool has_colors = HasColors();
    bool has_weights = HasWeights();
    utility::device_vector<Eigen::Vector2i> new_lines;
    utility::device_vector<float> new_weights;
    utility::device_vector<Eigen::Vector3f> new_colors;
    utility::device_vector<Eigen::Vector2i> sorted_edges = edges;
    thrust::sort(sorted_edges.begin(), sorted_edges.end());
    auto cnst_w = thrust::make_constant_iterator<float>(1.0);
    auto cnst_c = thrust::make_constant_iterator<Eigen::Vector3f>(Eigen::Vector3f::Ones());
    if (has_colors && has_weights) {
        auto func = tuple_element_compare_functor<EdgeWeightColor, 0, thrust::greater<Eigen::Vector2i>>();
        auto begin = make_tuple_begin(new_lines, new_weights, new_colors);
        auto end1 = thrust::set_difference(make_tuple_begin(lines_, edge_weights_, colors_),
                make_tuple_end(lines_, edge_weights_, colors_),
                make_tuple_iterator(sorted_edges.begin(), cnst_w, cnst_c),
                make_tuple_iterator(sorted_edges.end(), cnst_w, cnst_c),
                begin, func);
        resize_all(thrust::distance(begin, end1), new_lines, new_weights, new_colors);
        if (!is_directed_) {
            auto end2 = thrust::set_difference(make_tuple_begin(lines_, edge_weights_, colors_),
                    make_tuple_end(lines_, edge_weights_, colors_),
                    make_tuple_iterator(thrust::make_transform_iterator(sorted_edges.begin(), swap_index_functor<int>()), cnst_w, cnst_c),
                    make_tuple_iterator(thrust::make_transform_iterator(sorted_edges.end(), swap_index_functor<int>()), cnst_w, cnst_c),
                    begin, func);
            resize_all(thrust::distance(begin, end2), new_lines, new_weights, new_colors);
        }
    } else if (has_colors && !has_weights) {
        auto func = tuple_element_compare_functor<EdgeColor, 0, thrust::greater<Eigen::Vector2i>>();
        auto begin = make_tuple_begin(new_lines, new_colors);
        auto end1 = thrust::set_difference(make_tuple_begin(lines_, colors_),
                make_tuple_end(lines_, colors_),
                make_tuple_iterator(sorted_edges.begin(), cnst_c),
                make_tuple_iterator(sorted_edges.end(), cnst_c),
                begin, func);
        resize_all(thrust::distance(begin, end1), new_lines, new_colors);
        if (!is_directed_) {
            auto end2 = thrust::set_difference(make_tuple_begin(lines_, colors_),
                    make_tuple_end(lines_, colors_),
                    make_tuple_iterator(thrust::make_transform_iterator(sorted_edges.begin(), swap_index_functor<int>()), cnst_c),
                    make_tuple_iterator(thrust::make_transform_iterator(sorted_edges.end(), swap_index_functor<int>()), cnst_c),
                    begin, func);
            resize_all(thrust::distance(begin, end2), new_lines, new_colors);
        }
    } else if (!has_colors && has_weights) {
        auto func = tuple_element_compare_functor<EdgeWeight, 0, thrust::greater<Eigen::Vector2i>>();
        auto begin = make_tuple_begin(new_lines, new_weights);
        auto end1 = thrust::set_difference(make_tuple_begin(lines_, edge_weights_),
                make_tuple_end(lines_, edge_weights_),
                make_tuple_iterator(sorted_edges.begin(), cnst_w),
                make_tuple_iterator(sorted_edges.end(), cnst_w),
                begin, func);
        resize_all(thrust::distance(begin, end1), new_lines, new_weights);
        if (!is_directed_) {
            auto end2 = thrust::set_difference(make_tuple_begin(lines_, edge_weights_),
                    make_tuple_end(lines_, edge_weights_),
                    make_tuple_iterator(thrust::make_transform_iterator(sorted_edges.begin(), swap_index_functor<int>()), cnst_w),
                    make_tuple_iterator(thrust::make_transform_iterator(sorted_edges.end(), swap_index_functor<int>()), cnst_w),
                    begin, func);
            resize_all(thrust::distance(begin, end2), new_lines, new_weights);
        }
    } else {
        auto end1 = thrust::set_difference(lines_.begin(), lines_.end(),
                sorted_edges.begin(), sorted_edges.end(), new_lines.begin());
        new_lines.resize(thrust::distance(new_lines.begin(), end1));
        if (!is_directed_) {
            auto end2 = thrust::set_difference(lines_.begin(), lines_.end(),
                    thrust::make_transform_iterator(sorted_edges.begin(), swap_index_functor<int>()),
                    thrust::make_transform_iterator(sorted_edges.end(), swap_index_functor<int>()),
                    new_lines.begin());
            new_lines.resize(thrust::distance(new_lines.begin(), end2));
        }
    }
    thrust::swap(lines_, new_lines);
    thrust::swap(edge_weights_, new_weights);
    thrust::swap(colors_, new_colors);
    return ConstructGraph(false);
}

Graph &Graph::RemoveEdges(const thrust::host_vector<Eigen::Vector2i> &edges) {
    utility::device_vector<Eigen::Vector2i> d_edges = edges;
    return RemoveEdges(d_edges);
}

Graph &Graph::PaintEdgeColor(const Eigen::Vector2i &edge, const Eigen::Vector3f &color) {
    if (!HasColors()) {
        colors_.resize(lines_.size(), Eigen::Vector3f::Ones());
    }
    thrust::transform_if(colors_.begin(), colors_.end(), lines_.begin(), colors_.begin(),
                         [color] __device__ (const Eigen::Vector3f& c) { return color; },
                         [edge, is_directed = is_directed_] __device__ (const Eigen::Vector2i& line) {
                            return line == edge || (!is_directed && line == Eigen::Vector2i(edge[1], edge[0]));
                         });
    return *this;
}

Graph &Graph::PaintEdgesColor(const utility::device_vector<Eigen::Vector2i> &edges, const Eigen::Vector3f &color) {
    utility::device_vector<Eigen::Vector2i> sorted_edges = edges;
    utility::device_vector<size_t> indices(edges.size());
    thrust::sort(sorted_edges.begin(), sorted_edges.end());
    thrust::set_intersection(make_tuple_iterator(lines_.begin(), thrust::make_counting_iterator<size_t>(0)),
            make_tuple_iterator(lines_.end(), thrust::make_counting_iterator(lines_.size())),
            make_tuple_iterator(sorted_edges.begin(), thrust::make_constant_iterator<size_t>(0)),
            make_tuple_iterator(sorted_edges.end(), thrust::make_constant_iterator(sorted_edges.size())),
            make_tuple_iterator(thrust::make_discard_iterator(), indices.begin()),
            tuple_element_compare_functor<thrust::tuple<Eigen::Vector2i, size_t>, 0, thrust::greater<Eigen::Vector2i>>());
    thrust::for_each(thrust::make_permutation_iterator(colors_.begin(), indices.begin()),
                     thrust::make_permutation_iterator(colors_.begin(), indices.end()),
                     [color] __device__ (Eigen::Vector3f& c) { c = color; });
    if (!is_directed_) {
        swap_index(sorted_edges);
        thrust::sort(sorted_edges.begin(), sorted_edges.end());
        thrust::set_intersection(make_tuple_iterator(lines_.begin(), thrust::make_counting_iterator<size_t>(0)),
                make_tuple_iterator(lines_.end(), thrust::make_counting_iterator(lines_.size())),
                make_tuple_iterator(sorted_edges.begin(), thrust::make_constant_iterator<size_t>(0)),
                make_tuple_iterator(sorted_edges.end(), thrust::make_constant_iterator(sorted_edges.size())),
                make_tuple_iterator(thrust::make_discard_iterator(), indices.begin()),
                tuple_element_compare_functor<thrust::tuple<Eigen::Vector2i, size_t>, 0, thrust::greater<Eigen::Vector2i>>());
        thrust::for_each(thrust::make_permutation_iterator(colors_.begin(), indices.begin()),
                         thrust::make_permutation_iterator(colors_.begin(), indices.end()),
                         [color] __device__ (Eigen::Vector3f& c) { c = color; });
    }
    return *this;
}

Graph &Graph::PaintEdgesColor(const thrust::host_vector<Eigen::Vector2i> &edges, const Eigen::Vector3f &color) {
    utility::device_vector<Eigen::Vector2i> d_edges = edges;
    return PaintEdgesColor(d_edges, color);
}

Graph &Graph::PaintNodeColor(int node, const Eigen::Vector3f &color) {
    if (!HasNodeColors()) {
        node_colors_.resize(points_.size(), Eigen::Vector3f::Ones());
    }
    node_colors_[node] = color;
    return *this;
}

Graph &Graph::PaintNodesColor(const utility::device_vector<int> &nodes, const Eigen::Vector3f &color) {
    if (!HasNodeColors()) {
        node_colors_.resize(points_.size(), Eigen::Vector3f::Ones());
    }
    thrust::for_each(node_colors_.begin(), node_colors_.end(),
                     [color] __device__ (Eigen::Vector3f& c) { c = color; });
    return *this;
}

Graph &Graph::PaintNodesColor(const thrust::host_vector<int> &nodes, const Eigen::Vector3f &color) {
    utility::device_vector<int> d_nodes = nodes;
    return PaintNodesColor(d_nodes, color);
}

Graph &Graph::SetEdgeWeightsFromDistance() {
    edge_weights_.resize(lines_.size());
    Eigen::Vector3f *pt_ptr = thrust::raw_pointer_cast(points_.data());
    thrust::transform(lines_.begin(), lines_.end(), edge_weights_.begin(),
                      [pt_ptr] __device__ (const Eigen::Vector2i& edge) {
                          return (pt_ptr[edge[0]] - pt_ptr[edge[1]]).norm();
                      });
    return *this;
}

std::shared_ptr<Graph::SSSPResultArray> Graph::DijkstraPaths(int start_node_index, int end_node_index) const {
    auto out = std::make_shared<Graph::SSSPResultArray>();
    out->resize(points_.size());

    if (!IsConstructed()) {
        utility::LogError("[DijkstraPath] this graph is not constructed.");
        return out;
    }

    utility::device_vector<Eigen::Vector2i> sorted_lines = lines_;
    utility::device_vector<int> new_to_old_edge_table(lines_.size());
    utility::device_vector<int> old_to_new_edge_table(lines_.size());
    thrust::sequence(new_to_old_edge_table.begin(), new_to_old_edge_table.end(), 0);
    thrust::sort_by_key(sorted_lines.begin(), sorted_lines.end(), new_to_old_edge_table.begin(),
            [] __device__ (const Eigen::Vector2i &lhs, const Eigen::Vector2i &rhs) {
                return lhs[1] < rhs[1];
            });
    thrust::scatter(thrust::make_counting_iterator<size_t>(0), thrust::make_counting_iterator(lines_.size()),
            new_to_old_edge_table.begin(), old_to_new_edge_table.begin());
    utility::device_vector<int> open_flags(points_.size(), 0);
    utility::device_vector<size_t> indices(points_.size());
    thrust::sequence(indices.begin(), indices.end(), 0);
    SSSPResultArray res_tmp(lines_.size());
    SSSPResultArray res_tmp_s(points_.size());
    open_flags[start_node_index] = 1;
    (*out)[start_node_index] = SSSPResult(0.0, start_node_index);
    relax_functor func1(thrust::raw_pointer_cast(lines_.data()),
                        thrust::raw_pointer_cast(edge_index_offsets_.data()),
                        thrust::raw_pointer_cast(edge_weights_.data()),
                        thrust::raw_pointer_cast(old_to_new_edge_table.data()),
                        thrust::raw_pointer_cast(open_flags.data()),
                        thrust::raw_pointer_cast(out->data()),
                        thrust::raw_pointer_cast(res_tmp.data()));
    update_shortest_distances_functor func2(thrust::raw_pointer_cast(open_flags.data()),
                                            thrust::raw_pointer_cast(out->data()),
                                            thrust::raw_pointer_cast(res_tmp_s.data()));
    compare_path_length_functor func3(thrust::raw_pointer_cast(out->data()),
                                      thrust::raw_pointer_cast(open_flags.data()),
                                      end_node_index);
    size_t nt = points_.size();
    while (thrust::find(open_flags.begin(), open_flags.end(), 1) != open_flags.end()) {
        if (end_node_index >= 0 &&
            thrust::count_if(indices.begin(), indices.begin() + nt, func3) == 0) break;
        thrust::for_each(indices.begin(), indices.begin() + nt, func1);
        const auto begin = thrust::make_transform_iterator(sorted_lines.begin(), extract_element_functor<int, 2, 1>());
        auto end = thrust::reduce_by_key(begin, begin + sorted_lines.size(), res_tmp.begin(),
                                         indices.begin(), res_tmp_s.begin(),
                                         thrust::equal_to<int>(),
                                         [] __device__ (const SSSPResult& lhs, const SSSPResult& rhs) {
                                             return (lhs.shortest_distance_<= rhs.shortest_distance_) ? lhs : rhs;
                                         });
        nt = thrust::distance(indices.begin(), end.first);
        thrust::for_each(indices.begin(), indices.begin() + nt, func2);
    }
    return out;
}

std::shared_ptr<Graph::SSSPResultHostArray> Graph::DijkstraPathsHost(int start_node_index, int end_node_index) const {
    auto out = DijkstraPaths(start_node_index, end_node_index);
    auto h_out = std::make_shared<Graph::SSSPResultHostArray>();
    *h_out = *out;
    return h_out;
}

std::shared_ptr<thrust::host_vector<int>> Graph::DijkstraPath(int start_node_index, int end_node_index) const {
    auto res = DijkstraPaths(start_node_index, end_node_index);
    SSSPResultHostArray h_res = *res;
    auto path_nodes = std::make_shared<thrust::host_vector<int>>();
    if (h_res[end_node_index].prev_index_ < 0) return path_nodes;
    path_nodes->push_back(end_node_index);
    int prev_index = h_res[end_node_index].prev_index_;
    while (prev_index != start_node_index) {
        path_nodes->push_back(prev_index);
        prev_index = h_res[prev_index].prev_index_;
    }
    path_nodes->push_back(start_node_index);
    thrust::reverse(path_nodes->begin(), path_nodes->end());
    return path_nodes;
}

}
}