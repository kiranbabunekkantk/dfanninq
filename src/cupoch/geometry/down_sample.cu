#include "hip/hip_runtime.h"
#include "cupoch/geometry/pointcloud.h"
#include "cupoch/utility/console.h"
#include "cupoch/utility/helper.h"
#include <thrust/gather.h>


using namespace cupoch;
using namespace cupoch::geometry;

namespace {

struct compute_key_functor {
    compute_key_functor(const Eigen::Vector3f& voxel_min_bound, float voxel_size)
        : voxel_min_bound_(voxel_min_bound), voxel_size_(voxel_size) {};
    const Eigen::Vector3f voxel_min_bound_;
    const float voxel_size_;
    __device__
    Eigen::Vector3i operator()(const Eigen::Vector3f_u& pt) {
        auto ref_coord = (pt - voxel_min_bound_) / voxel_size_;
        return Eigen::Vector3i(int(floor(ref_coord(0))), int(floor(ref_coord(1))), int(floor(ref_coord(2))));
    }
};

template<typename OutputIterator, class... Args>
__host__
int CalcAverageByKey(thrust::device_vector<Eigen::Vector3i>& keys,
                     OutputIterator buf_begins, OutputIterator output_begins) {
    const size_t n = keys.size();
    thrust::sort_by_key(keys.begin(), keys.end(), buf_begins);

    thrust::device_vector<Eigen::Vector3i> keys_out(n);
    thrust::device_vector<int> counts(n);
    auto end1 = thrust::reduce_by_key(keys.begin(), keys.end(),
                                      thrust::make_constant_iterator(1),
                                      keys_out.begin(), counts.begin());
    int n_out = thrust::distance(counts.begin(), end1.second);
    counts.resize(n_out);

    thrust::equal_to<Eigen::Vector3i> binary_pred;
    add_tuple_functor<Args...> add_func;
    auto end2 = thrust::reduce_by_key(keys.begin(), keys.end(), buf_begins,
                                      keys_out.begin(), output_begins,
                                      binary_pred, add_func);

    devided_tuple_functor<Args...> dv_func;
    thrust::transform(output_begins, output_begins + n_out,
                      counts.begin(), output_begins,
                      dv_func);
    return n_out;
}

}

std::shared_ptr<PointCloud> PointCloud::SelectDownSample(const thrust::device_vector<size_t> &indices, bool invert) const {
    auto output = std::make_shared<PointCloud>();
    const bool has_normals = HasNormals();
    const bool has_colors = HasColors();

    output->points_.resize(indices.size());
    thrust::gather(indices.begin(), indices.end(), points_.begin(), output->points_.begin());
    if (HasNormals()) {
        output->normals_.resize(indices.size());
        thrust::gather(indices.begin(), indices.end(), normals_.begin(), output->normals_.begin());
    }
    if (HasColors()) {
        output->colors_.resize(indices.size());
        thrust::gather(indices.begin(), indices.end(), colors_.begin(), output->colors_.begin());
    }
    return output;
}

std::shared_ptr<PointCloud> PointCloud::VoxelDownSample(float voxel_size) const {
    auto output = std::make_shared<PointCloud>();
    if (voxel_size <= 0.0) {
        utility::LogWarning("[VoxelDownSample] voxel_size <= 0.\n");
        return output;
    }

    const Eigen::Vector3f voxel_size3 = Eigen::Vector3f(voxel_size, voxel_size, voxel_size);
    const Eigen::Vector3f voxel_min_bound = GetMinBound() - voxel_size3 * 0.5;
    const Eigen::Vector3f voxel_max_bound = GetMaxBound() + voxel_size3 * 0.5;

    if (voxel_size * std::numeric_limits<int>::max() < (voxel_max_bound - voxel_min_bound).maxCoeff()) {
        utility::LogWarning("[VoxelDownSample] voxel_size is too small.\n");
        return output;
    }

    const int n = points_.size();
    const bool has_normals = HasNormals();
    const bool has_colors = HasColors();
    compute_key_functor ck_func(voxel_min_bound, voxel_size);
    thrust::device_vector<Eigen::Vector3i> keys(n);
    thrust::transform(points_.begin(), points_.end(), keys.begin(), ck_func);

    thrust::device_vector<Eigen::Vector3f_u> sorted_points = points_;
    output->points_.resize(n);
    if (!has_normals && !has_colors) {
        typedef thrust::tuple<thrust::device_vector<Eigen::Vector3f_u>::iterator> IteratorTuple;
        typedef thrust::zip_iterator<IteratorTuple> ZipIterator;
        auto n_out = CalcAverageByKey<ZipIterator, Eigen::Vector3f_u>(keys,
                    thrust::make_zip_iterator(thrust::make_tuple(sorted_points.begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(output->points_.begin())));
        output->points_.resize(n_out);
    } else if (has_normals && !has_colors) {
        thrust::device_vector<Eigen::Vector3f_u> sorted_normals = normals_;
        output->normals_.resize(n);
        typedef thrust::tuple<thrust::device_vector<Eigen::Vector3f_u>::iterator, thrust::device_vector<Eigen::Vector3f_u>::iterator> IteratorTuple;
        typedef thrust::zip_iterator<IteratorTuple> ZipIterator;
        auto n_out = CalcAverageByKey<ZipIterator, Eigen::Vector3f_u, Eigen::Vector3f_u>(keys,
                    thrust::make_zip_iterator(thrust::make_tuple(sorted_points.begin(), sorted_normals.begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(output->points_.begin(), output->normals_.begin())));
        output->points_.resize(n_out);
        output->normals_.resize(n_out);
        thrust::for_each(output->normals_.begin(), output->normals_.end(), [] __device__ (Eigen::Vector3f_u& nl) {nl.normalize();});
    } else if (!has_normals && has_colors) {
        thrust::device_vector<Eigen::Vector3f_u> sorted_colors = colors_;
        output->colors_.resize(n);
        typedef thrust::tuple<thrust::device_vector<Eigen::Vector3f_u>::iterator, thrust::device_vector<Eigen::Vector3f_u>::iterator> IteratorTuple;
        typedef thrust::zip_iterator<IteratorTuple> ZipIterator;
        auto n_out = CalcAverageByKey<ZipIterator, Eigen::Vector3f_u, Eigen::Vector3f_u>(keys,
                    thrust::make_zip_iterator(thrust::make_tuple(sorted_points.begin(), sorted_colors.begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(output->points_.begin(), output->colors_.begin())));
        output->points_.resize(n_out);
        output->colors_.resize(n_out);
    } else {
        thrust::device_vector<Eigen::Vector3f_u> sorted_normals = normals_;
        thrust::device_vector<Eigen::Vector3f_u> sorted_colors = colors_;
        output->normals_.resize(n);
        output->colors_.resize(n);
        typedef thrust::tuple<thrust::device_vector<Eigen::Vector3f_u>::iterator, thrust::device_vector<Eigen::Vector3f_u>::iterator, thrust::device_vector<Eigen::Vector3f_u>::iterator> IteratorTuple;
        typedef thrust::zip_iterator<IteratorTuple> ZipIterator;
        auto n_out = CalcAverageByKey<ZipIterator, Eigen::Vector3f_u, Eigen::Vector3f_u, Eigen::Vector3f_u>(keys,
                    thrust::make_zip_iterator(thrust::make_tuple(sorted_points.begin(), sorted_normals.begin(), sorted_colors.begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(output->points_.begin(), output->normals_.begin(), output->colors_.begin())));
        output->points_.resize(n_out);
        output->normals_.resize(n_out);
        output->colors_.resize(n_out);
        thrust::for_each(output->normals_.begin(), output->normals_.end(), [] __device__ (Eigen::Vector3f_u& nl) {nl.normalize();});
    }

    utility::LogDebug(
            "Pointcloud down sampled from {:d} points to {:d} points.\n",
            (int)points_.size(), (int)output->points_.size());
    return output;
}
