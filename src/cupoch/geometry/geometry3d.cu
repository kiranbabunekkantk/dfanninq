#include "hip/hip_runtime.h"
#include "cupoch/geometry/geometry3d.h"
#include "cupoch/utility/console.h"

using namespace cupoch;
using namespace cupoch::geometry;

namespace {

struct elementwise_min_functor {
    __device__
    Eigen::Vector3f operator()(const Eigen::Vector3f& a, const Eigen::Vector3f& b) {
        return a.array().min(b.array()).matrix();
    }
};
    
struct elementwise_max_functor {
    __device__
    Eigen::Vector3f operator()(const Eigen::Vector3f& a, const Eigen::Vector3f& b) {
        return a.array().max(b.array()).matrix();
    }
};

struct transform_points_functor {
    transform_points_functor(const Eigen::Matrix4f& transform) : transform_(transform){};
    const Eigen::Matrix4f transform_;
    __device__
    void operator()(Eigen::Vector3f& pt) {
        const Eigen::Vector4f new_pt = transform_ * Eigen::Vector4f(pt(0), pt(1), pt(2), 1.0);
        pt = new_pt.head<3>() / new_pt(3);
    }
};

struct transform_normals_functor {
    transform_normals_functor(const Eigen::Matrix4f& transform) : transform_(transform){};
    const Eigen::Matrix4f transform_;
    __device__
    void operator()(Eigen::Vector3f& nl) {
        const Eigen::Vector4f new_pt = transform_ * Eigen::Vector4f(nl(0), nl(1), nl(2), 0.0);
        nl = new_pt.head<3>();
    }
};
}

Eigen::Vector3f Geometry3D::ComputeMinBound(const thrust::device_vector<Eigen::Vector3f>& points) const {
    if (points.empty()) return Eigen::Vector3f::Zero();
    Eigen::Vector3f init = points[0];
    return thrust::reduce(points.begin(), points.end(), init, elementwise_min_functor());
}

Eigen::Vector3f Geometry3D::ComputeMaxBound(const thrust::device_vector<Eigen::Vector3f>& points) const {
    if (points.empty()) return Eigen::Vector3f::Zero();
    Eigen::Vector3f init = points[0];
    return thrust::reduce(points.begin(), points.end(), init, elementwise_max_functor());
}

Eigen::Vector3f Geometry3D::ComuteCenter(const thrust::device_vector<Eigen::Vector3f>& points) const {
    Eigen::Vector3f init = Eigen::Vector3f::Zero();
    if (points.empty()) return init;
    Eigen::Vector3f sum = thrust::reduce(points.begin(), points.end(), init, thrust::plus<Eigen::Vector3f>());
    return sum / points.size();
}

void Geometry3D::ResizeAndPaintUniformColor(thrust::device_vector<Eigen::Vector3f>& colors,
    const size_t size,
    const Eigen::Vector3f& color) {
    colors.resize(size);
    Eigen::Vector3f clipped_color = color;
    if (color.minCoeff() < 0 || color.maxCoeff() > 1) {
        utility::LogWarning(
                "invalid color in PaintUniformColor, clipping to [0, 1]");
        clipped_color = clipped_color.array()
                                .max(Eigen::Vector3f(0, 0, 0).array())
                                .matrix();
        clipped_color = clipped_color.array()
                                .min(Eigen::Vector3f(1, 1, 1).array())
                                .matrix();
    }
    thrust::fill(colors.begin(), colors.end(), clipped_color);
}

void Geometry3D::TransformPoints(const Eigen::Matrix4f& transformation,
                                 thrust::device_vector<Eigen::Vector3f>& points) {
    transform_points_functor func(transformation);
    thrust::for_each(points.begin(), points.end(), func);
}

void Geometry3D::TransformPoints(hipStream_t stream, const Eigen::Matrix4f& transformation,
                                 thrust::device_vector<Eigen::Vector3f>& points) {
    transform_points_functor func(transformation);
    thrust::for_each(thrust::cuda::par.on(stream), points.begin(), points.end(), func);
}

void Geometry3D::TransformNormals(const Eigen::Matrix4f& transformation,
                                  thrust::device_vector<Eigen::Vector3f>& normals) {
    transform_normals_functor func(transformation);
    thrust::for_each(normals.begin(), normals.end(), func);
}

void Geometry3D::TransformNormals(hipStream_t stream, const Eigen::Matrix4f& transformation,
                                  thrust::device_vector<Eigen::Vector3f>& normals) {
    transform_normals_functor func(transformation);
    thrust::for_each(thrust::cuda::par.on(stream), normals.begin(), normals.end(), func);
}