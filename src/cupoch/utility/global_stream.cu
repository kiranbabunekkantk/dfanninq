#include "cupoch/utility/global_stream.h"
#include <mutex>

using namespace cupoch;
using namespace cupoch::utility;

hipStream_t cupoch::utility::GetGlobalStream(size_t i) {
    static std::once_flag streamInitFlags[MAX_DEVICES];
    static hipStream_t streams[MAX_DEVICES];
    std::call_once(streamInitFlags[i], [i]() {
        hipStreamCreate(&(streams[i]));
    });
    return streams[i];
}