#include "cupoch/utility/platform.h"
#include <mutex>

using namespace cupoch;
using namespace cupoch::utility;

hipStream_t cupoch::utility::GetStream(size_t i) {
    static std::once_flag streamInitFlags[MAX_DEVICES];
    static hipStream_t streams[MAX_DEVICES];
    std::call_once(streamInitFlags[i], [i]() {
        hipStreamCreate(&(streams[i]));
    });
    return streams[i];
}

int cupoch::utility::GetDevice() {
    int device_no;
    hipGetDevice(&device_no);
    return device_no;
}

void cupoch::utility::SetDevice(int device_no) {
    hipSetDevice(device_no);
}