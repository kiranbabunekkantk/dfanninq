#include "cupoch/io/class_io/trianglemesh_io.h"

using namespace cupoch;
using namespace cupoch::io;

void HostTriangleMesh::FromDevice(const geometry::TriangleMesh& trianglemesh) {
    vertices_.resize(trianglemesh.vertices_.size());
    vertex_normals_.resize(trianglemesh.vertex_normals_.size());
    vertex_colors_.resize(trianglemesh.vertex_colors_.size());
    utility::CopyFromDeviceMultiStream(trianglemesh.vertices_, vertices_);
    utility::CopyFromDeviceMultiStream(trianglemesh.vertex_normals_, vertex_normals_);
    utility::CopyFromDeviceMultiStream(trianglemesh.vertex_colors_, vertex_colors_);
    utility::CopyFromDeviceMultiStream(trianglemesh.triangles_, triangles_);
    utility::CopyFromDeviceMultiStream(trianglemesh.triangle_normals_, triangle_normals_);
    utility::CopyFromDeviceMultiStream(trianglemesh.triangle_uvs_, triangle_uvs_);
    hipDeviceSynchronize();
}

void HostTriangleMesh::ToDevice(geometry::TriangleMesh& trianglemesh) const {
    trianglemesh.vertices_.resize(vertices_.size());
    trianglemesh.vertex_normals_.resize(vertex_normals_.size());
    trianglemesh.vertex_colors_.resize(vertex_colors_.size());
    utility::CopyToDeviceMultiStream(vertices_, trianglemesh.vertices_);
    utility::CopyToDeviceMultiStream(vertex_normals_, trianglemesh.vertex_normals_);
    utility::CopyToDeviceMultiStream(vertex_colors_, trianglemesh.vertex_colors_);
    utility::CopyToDeviceMultiStream(triangles_, trianglemesh.triangles_);
    utility::CopyToDeviceMultiStream(triangle_normals_, trianglemesh.triangle_normals_);
    utility::CopyToDeviceMultiStream(triangle_uvs_, trianglemesh.triangle_uvs_);
    hipDeviceSynchronize();
}

void HostTriangleMesh::Clear() {
    vertices_.clear();
    vertex_normals_.clear();
    vertex_colors_.clear();
}