#include "hip/hip_runtime.h"
#include "cupoc/registration/registration.h"
#include "cupoc/geometry/kdtree_flann.h"
#include "cupoc/utility/helper.h"
#include "cupoc/utility/console.h"

using namespace cupoc;
using namespace cupoc::registration;

namespace {

struct extact_knn_distance_functor {
    __device__
    float operator() (const geometry::KNNDistances& x) const {
        return max(x[0], 0.0);
    }
};

struct make_correspondence_pair_functor {
   __device__
   thrust::tuple<int, int> operator() (int i, const geometry::KNNIndices& idxs) const {
        if (idxs[0] < 0) {
            return thrust::make_tuple(-1, -1);
        } else {
            return thrust::make_tuple(i, idxs[0]);
        }
   }
};

RegistrationResult GetRegistrationResultAndCorrespondences(
    const geometry::PointCloud &source,
    const geometry::PointCloud &target,
    const geometry::KDTreeFlann &target_kdtree,
    float max_correspondence_distance,
    const Eigen::Matrix4f &transformation) {
    RegistrationResult result(transformation);
    if (max_correspondence_distance <= 0.0) {
        return result;
    }

    const int n_pt = source.points_.size();
    thrust::device_vector<geometry::KNNIndices> indices(n_pt);
    thrust::device_vector<geometry::KNNDistances> dists(n_pt);
    target_kdtree.SearchHybrid(source.points_, max_correspondence_distance,
                               1, indices, dists);
    const float error2 = thrust::transform_reduce(dists.begin(), dists.end(),
                                                  extact_knn_distance_functor(),
                                                  0.0f, thrust::plus<float>());
    result.correspondence_set_.resize(n_pt);
    thrust::transform(thrust::make_counting_iterator(0), thrust::make_counting_iterator(n_pt),
                      indices.begin(), result.correspondence_set_.begin(),
                      make_correspondence_pair_functor());
    auto end = thrust::remove_if(result.correspondence_set_.begin(), result.correspondence_set_.end(),
                                 [] __device__ (const thrust::tuple<int, int>& x) -> bool {return (thrust::get<0>(x) < 0);});
    int n_out = static_cast<int>(end - result.correspondence_set_.begin());
    result.correspondence_set_.resize(n_out);

    if (result.correspondence_set_.empty()) {
        result.fitness_ = 0.0;
        result.inlier_rmse_ = 0.0;
    } else {
        size_t corres_number = result.correspondence_set_.size();
        result.fitness_ = (float)corres_number / (float)source.points_.size();
        result.inlier_rmse_ = std::sqrt(error2 / (float)corres_number);
    }
    return result;
}

}

RegistrationResult cupoc::registration::RegistrationICP(
    const geometry::PointCloud &source,
    const geometry::PointCloud &target,
    float max_correspondence_distance,
    const Eigen::Matrix4f &init /* = Eigen::Matrix4f::Identity()*/,
    const TransformationEstimation &estimation
    /* = TransformationEstimationPointToPoint(false)*/,
    const ICPConvergenceCriteria
            &criteria /* = ICPConvergenceCriteria()*/) {
    if (max_correspondence_distance <= 0.0) {
        utility::LogError("Invalid max_correspondence_distance.");
    }

    if ((estimation.GetTransformationEstimationType() ==
                TransformationEstimationType::PointToPlane ||
         estimation.GetTransformationEstimationType() ==
                TransformationEstimationType::ColoredICP) &&
        (!source.HasNormals() || !target.HasNormals())) {
        utility::LogError(
                "TransformationEstimationPointToPlane and "
                "TransformationEstimationColoredICP "
                "require pre-computed normal vectors.");
    }

    Eigen::Matrix4f transformation = init;
    geometry::KDTreeFlann kdtree(target);
    geometry::PointCloud pcd = source;
    if (init.isIdentity() == false) {
        pcd.Transform(init);
    }
    RegistrationResult result;
    result = GetRegistrationResultAndCorrespondences(
            pcd, target, kdtree, max_correspondence_distance, transformation);
    for (int i = 0; i < criteria.max_iteration_; i++) {
        utility::LogDebug("ICP Iteration #{:d}: Fitness {:.4f}, RMSE {:.4f}", i,
                          result.fitness_, result.inlier_rmse_);
        Eigen::Matrix4f update = estimation.ComputeTransformation(
                pcd, target, result.correspondence_set_);
        transformation = update * transformation;
        pcd.Transform(update);
        RegistrationResult backup = result;
        result = GetRegistrationResultAndCorrespondences(
                pcd, target, kdtree, max_correspondence_distance,
                transformation);
        if (std::abs(backup.fitness_ - result.fitness_) <
                    criteria.relative_fitness_ &&
            std::abs(backup.inlier_rmse_ - result.inlier_rmse_) <
                    criteria.relative_rmse_) {
            break;
        }
    }
    return result;
}