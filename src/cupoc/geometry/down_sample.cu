#include "hip/hip_runtime.h"
#include "cupoc/geometry/pointcloud.h"
#include "cupoc/geometry/geometry3d.h"
#include "cupoc/utility/console.h"
#include "cupoc/utility/helper.h"
#include <thrust/gather.h>


using namespace cupoc;
using namespace cupoc::geometry;

namespace {

struct compute_key_functor {
    compute_key_functor(const Eigen::Vector3f& voxel_min_bound, float voxel_size)
        : voxel_min_bound_(voxel_min_bound), voxel_size_(voxel_size) {};
    const Eigen::Vector3f& voxel_min_bound_;
    const float voxel_size_;
    __device__
    Eigen::Vector3i operator()(const Eigen::Vector3f_u& pt) {
        auto ref_coord = (pt - voxel_min_bound_) / voxel_size_;
        return Eigen::Vector3i(int(floor(ref_coord(0))), int(floor(ref_coord(1))), int(floor(ref_coord(2))));
    }
};

template<typename OutputIterator, class... Args>
__host__
int CalcAverageByKey(thrust::device_vector<Eigen::Vector3i>& keys,
                     OutputIterator buf_begins, OutputIterator output_begins) {
    const size_t n = keys.size();
    thrust::sort_by_key(keys.begin(), keys.end(), buf_begins);

    thrust::device_vector<Eigen::Vector3i> keys_out(n);
    thrust::device_vector<int> counts(n);
    auto end1 = thrust::reduce_by_key(keys.begin(), keys.end(),
                                      thrust::make_constant_iterator(1),
                                      keys_out.begin(), counts.begin());
    int n_out = static_cast<int>(end1.second - counts.begin());
    counts.resize(n_out);

    thrust::equal_to<Eigen::Vector3i> binary_pred;
    add_tuple_functor<Args...> add_func;
    auto end2 = thrust::reduce_by_key(keys.begin(), keys.end(), buf_begins,
                                      keys_out.begin(), output_begins,
                                      binary_pred, add_func);

    devided_tuple_functor<Args...> dv_func;
    thrust::transform(output_begins, output_begins + n_out,
                      counts.begin(), output_begins,
                      dv_func);
    return n_out;
}

}

utility::shared_ptr<PointCloud> PointCloud::SelectDownSample(const thrust::device_vector<size_t> &indices, bool invert) const {
    auto output = utility::shared_ptr<PointCloud>(new PointCloud());
    const bool has_normals = HasNormals();
    const bool has_colors = HasColors();

    output->points_.resize(indices.size());
    thrust::gather(indices.begin(), indices.end(), points_.begin(), output->points_.begin());
    return output;
}

utility::shared_ptr<PointCloud> PointCloud::VoxelDownSample(float voxel_size) const {
    auto output = utility::shared_ptr<PointCloud>(new PointCloud());
    if (voxel_size <= 0.0) {
        utility::LogWarning("[VoxelDownSample] voxel_size <= 0.\n");
        return output;
    }

    const Eigen::Vector3f voxel_size3 = Eigen::Vector3f(voxel_size, voxel_size, voxel_size);
    const Eigen::Vector3f voxel_min_bound = GetMinBound() - voxel_size3 * 0.5;
    const Eigen::Vector3f voxel_max_bound = GetMaxBound() + voxel_size3 * 0.5;

    if (voxel_size * std::numeric_limits<int>::max() < (voxel_max_bound - voxel_min_bound).maxCoeff()) {
        utility::LogWarning("[VoxelDownSample] voxel_size is too small.\n");
        return output;
    }

    const int n = points_.size();
    const bool has_normals = HasNormals();
    const bool has_colors = HasColors();
    compute_key_functor ck_func(voxel_min_bound, voxel_size);
    thrust::device_vector<Eigen::Vector3i> keys(n);
    thrust::transform(points_.begin(), points_.end(), keys.begin(), ck_func);

    thrust::device_vector<Eigen::Vector3f_u> sorted_points = points_;
    output->points_.resize(n);
    if (!has_normals && !has_colors) {
        typedef thrust::tuple<thrust::device_vector<Eigen::Vector3f_u>::iterator> IteratorTuple;
        typedef thrust::zip_iterator<IteratorTuple> ZipIterator;
        auto n_out = CalcAverageByKey<ZipIterator, Eigen::Vector3f_u>(keys,
                    thrust::make_zip_iterator(thrust::make_tuple(sorted_points.begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(output->points_.begin())));
        output->points_.resize(n_out);
    } else if (has_normals && !has_colors) {
        thrust::device_vector<Eigen::Vector3f_u> sorted_normals = normals_;
        output->normals_.resize(n);
        typedef thrust::tuple<thrust::device_vector<Eigen::Vector3f_u>::iterator, thrust::device_vector<Eigen::Vector3f_u>::iterator> IteratorTuple;
        typedef thrust::zip_iterator<IteratorTuple> ZipIterator;
        auto n_out = CalcAverageByKey<ZipIterator, Eigen::Vector3f_u, Eigen::Vector3f_u>(keys,
                    thrust::make_zip_iterator(thrust::make_tuple(sorted_points.begin(), sorted_normals.begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(output->points_.begin(), output->normals_.begin())));
        output->points_.resize(n_out);
        output->normals_.resize(n_out);
    } else if (!has_normals && has_colors) {
        thrust::device_vector<Eigen::Vector3f_u> sorted_colors = colors_;
        output->colors_.resize(n);
        typedef thrust::tuple<thrust::device_vector<Eigen::Vector3f_u>::iterator, thrust::device_vector<Eigen::Vector3f_u>::iterator> IteratorTuple;
        typedef thrust::zip_iterator<IteratorTuple> ZipIterator;
        auto n_out = CalcAverageByKey<ZipIterator, Eigen::Vector3f_u, Eigen::Vector3f_u>(keys,
                    thrust::make_zip_iterator(thrust::make_tuple(sorted_points.begin(), sorted_colors.begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(output->points_.begin(), output->colors_.begin())));
        output->points_.resize(n_out);
        output->colors_.resize(n_out);
    } else {
        thrust::device_vector<Eigen::Vector3f_u> sorted_normals = normals_;
        thrust::device_vector<Eigen::Vector3f_u> sorted_colors = colors_;
        output->normals_.resize(n);
        output->colors_.resize(n);
        typedef thrust::tuple<thrust::device_vector<Eigen::Vector3f_u>::iterator, thrust::device_vector<Eigen::Vector3f_u>::iterator, thrust::device_vector<Eigen::Vector3f_u>::iterator> IteratorTuple;
        typedef thrust::zip_iterator<IteratorTuple> ZipIterator;
        auto n_out = CalcAverageByKey<ZipIterator, Eigen::Vector3f_u, Eigen::Vector3f_u, Eigen::Vector3f_u>(keys,
                    thrust::make_zip_iterator(thrust::make_tuple(sorted_points.begin(), sorted_normals.begin(), sorted_colors.begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(output->points_.begin(), output->normals_.begin(), output->colors_.begin())));
        output->points_.resize(n_out);
        output->normals_.resize(n_out);
        output->colors_.resize(n_out);
    }

    utility::LogDebug(
            "Pointcloud down sampled from {:d} points to {:d} points.\n",
            (int)points_.size(), (int)output->points_.size());
    return output;
}
