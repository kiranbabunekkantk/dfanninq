#include "hip/hip_runtime.h"
#include <Eigen/Geometry>
#include "cupoc/geometry/pointcloud.h"
#include "cupoc/geometry/kdtree_flann.h"
#include "cupoc/utility/console.h"

using namespace cupoc;
using namespace cupoc::geometry;

namespace {

__device__
Eigen::Vector3f ComputeEigenvector0(const Eigen::Matrix3f &A, float eval0) {
    Eigen::Vector3f row0(A(0, 0) - eval0, A(0, 1), A(0, 2));
    Eigen::Vector3f row1(A(0, 1), A(1, 1) - eval0, A(1, 2));
    Eigen::Vector3f row2(A(0, 2), A(1, 2), A(2, 2) - eval0);
    Eigen::Vector3f r0xr1 = row0.cross(row1);
    Eigen::Vector3f r0xr2 = row0.cross(row2);
    Eigen::Vector3f r1xr2 = row1.cross(row2);
    float d0 = r0xr1.dot(r0xr1);
    float d1 = r0xr2.dot(r0xr2);
    float d2 = r1xr2.dot(r1xr2);

    float dmax = d0;
    int imax = 0;
    if (d1 > dmax) {
        dmax = d1;
        imax = 1;
    }
    if (d2 > dmax) {
        imax = 2;
    }

    if (imax == 0) {
        return r0xr1 / std::sqrt(d0);
    } else if (imax == 1) {
        return r0xr2 / std::sqrt(d1);
    } else {
        return r1xr2 / std::sqrt(d2);
    }
}
    
__device__
Eigen::Vector3f ComputeEigenvector1(const Eigen::Matrix3f &A,
                                    const Eigen::Vector3f &evec0,
                                    double eval1) {
    Eigen::Vector3f U, V;
    if (std::abs(evec0(0)) > std::abs(evec0(1))) {
        float inv_length =
                1 / std::sqrt(evec0(0) * evec0(0) + evec0(2) * evec0(2));
        U << -evec0(2) * inv_length, 0, evec0(0) * inv_length;
    } else {
        float inv_length =
                1 / std::sqrt(evec0(1) * evec0(1) + evec0(2) * evec0(2));
        U << 0, evec0(2) * inv_length, -evec0(1) * inv_length;
    }
    V = evec0.cross(U);

    Eigen::Vector3f AU(A(0, 0) * U(0) + A(0, 1) * U(1) + A(0, 2) * U(2),
                      A(0, 1) * U(0) + A(1, 1) * U(1) + A(1, 2) * U(2),
                      A(0, 2) * U(0) + A(1, 2) * U(1) + A(2, 2) * U(2));

    Eigen::Vector3f AV = {A(0, 0) * V(0) + A(0, 1) * V(1) + A(0, 2) * V(2),
                          A(0, 1) * V(0) + A(1, 1) * V(1) + A(1, 2) * V(2),
                          A(0, 2) * V(0) + A(1, 2) * V(1) + A(2, 2) * V(2)};
    
    float m00 = U(0) * AU(0) + U(1) * AU(1) + U(2) * AU(2) - eval1;
    float m01 = U(0) * AV(0) + U(1) * AV(1) + U(2) * AV(2);
    float m11 = V(0) * AV(0) + V(1) * AV(1) + V(2) * AV(2) - eval1;
    
    float absM00 = std::abs(m00);
    float absM01 = std::abs(m01);
    float absM11 = std::abs(m11);
    float max_abs_comp;
    if (absM00 >= absM11) {
        max_abs_comp = max(absM00, absM01);
        if (max_abs_comp > 0) {
            if (absM00 >= absM01) {
                m01 /= m00;
                m00 = 1 / std::sqrt(1 + m01 * m01);
                m01 *= m00;
            } else {
                m00 /= m01;
                m01 = 1 / std::sqrt(1 + m00 * m00);
                m00 *= m01;
            }
            return m01 * U - m00 * V;
        } else {
            return U;
        }
    } else {
        max_abs_comp = max(absM11, absM01);
        if (max_abs_comp > 0) {
            if (absM11 >= absM01) {
                m01 /= m11;
                m11 = 1 / std::sqrt(1 + m01 * m01);
                m01 *= m11;
            } else {
                m11 /= m01;
                m01 = 1 / std::sqrt(1 + m11 * m11);
                m11 *= m01;
            }
            return m11 * U - m01 * V;
        } else {
            return U;
        }
    }
}

__device__
Eigen::Vector3f FastEigen3x3(Eigen::Matrix3f &A) {
    // Previous version based on:
    // https://en.wikipedia.org/wiki/Eigenvalue_algorithm#3.C3.973_matrices
    // Current version based on
    // https://www.geometrictools.com/Documentation/RobustEigenSymmetric3x3.pdf
    // which handles edge cases like points on a plane

    float max_coeff = A.maxCoeff();
    if (max_coeff == 0) {
        return Eigen::Vector3f::Zero();
    }
    A /= max_coeff;

    float norm = A(0, 1) * A(0, 1) + A(0, 2) * A(0, 2) + A(1, 2) * A(1, 2);
    if (norm > 0) {
        Eigen::Vector3f eval;
        Eigen::Vector3f evec0;
        Eigen::Vector3f evec1;
        Eigen::Vector3f evec2;

        float q = (A(0, 0) + A(1, 1) + A(2, 2)) / 3;

        float b00 = A(0, 0) - q;
        float b11 = A(1, 1) - q;
        float b22 = A(2, 2) - q;

        float p =
                std::sqrt((b00 * b00 + b11 * b11 + b22 * b22 + norm * 2) / 6);

        float c00 = b11 * b22 - A(1, 2) * A(1, 2);
        float c01 = A(0, 1) * b22 - A(1, 2) * A(0, 2);
        float c02 = A(0, 1) * A(1, 2) - b11 * A(0, 2);
        float det = (b00 * c00 - A(0, 1) * c01 + A(0, 2) * c02) / (p * p * p);

        float half_det = det * 0.5;
        half_det = min(max(half_det, -1.0), 1.0);

        float angle = std::acos(half_det) / (float)3;
        float const two_thirds_pi = 2.09439510239319549;
        float beta2 = std::cos(angle) * 2;
        float beta0 = std::cos(angle + two_thirds_pi) * 2;
        float beta1 = -(beta0 + beta2);

        eval(0) = q + p * beta0;
        eval(1) = q + p * beta1;
        eval(2) = q + p * beta2;

        if (half_det >= 0) {
            evec2 = ComputeEigenvector0(A, eval(2));
            if (eval(2) < eval(0) && eval(2) < eval(1)) {
                A *= max_coeff;
                return evec2;
            }
            evec1 = ComputeEigenvector1(A, evec2, eval(1));
            A *= max_coeff;
            if (eval(1) < eval(0) && eval(1) < eval(2)) {
                return evec1;
            }
            evec0 = evec1.cross(evec2);
            return evec0;
        } else {
            evec0 = ComputeEigenvector0(A, eval(0));
            if (eval(0) < eval(1) && eval(0) < eval(2)) {
                A *= max_coeff;
                return evec0;
            }
            evec1 = ComputeEigenvector1(A, evec0, eval(1));
            A *= max_coeff;
            if (eval(1) < eval(0) && eval(1) < eval(2)) {
                return evec1;
            }
            evec2 = evec0.cross(evec1);
            return evec2;
        }
    } else {
        A *= max_coeff;
        if (A(0, 0) < A(1, 1) && A(0, 0) < A(2, 2)) {
            return Eigen::Vector3f(1, 0, 0);
        } else if (A(1, 1) < A(0, 0) && A(1, 1) < A(2, 2)) {
            return Eigen::Vector3f(0, 1, 0);
        } else {
            return Eigen::Vector3f(0, 0, 1);
        }
    }
}

__device__
Eigen::Vector3f_u ComputeNormal(const Eigen::Vector3f_u* points,
                                const KNNIndices &indices, int knn) {
    if (indices[0] < 0) return Eigen::Vector3f_u(0.0, 0.0, 1.0);

    Eigen::Matrix3f covariance;
    Eigen::Matrix<float, 9, 1> cumulants;
    cumulants.setZero();
    for (size_t i = 0; i < knn; i++) {
        const Eigen::Vector3f_u& point = points[indices[i]];
        cumulants(0) += point(0);
        cumulants(1) += point(1);
        cumulants(2) += point(2);
        cumulants(3) += point(0) * point(0);
        cumulants(4) += point(0) * point(1);
        cumulants(5) += point(0) * point(2);
        cumulants(6) += point(1) * point(1);
        cumulants(7) += point(1) * point(2);
        cumulants(8) += point(2) * point(2);
    }
    cumulants /= (double)indices.size();
    covariance(0, 0) = cumulants(3) - cumulants(0) * cumulants(0);
    covariance(1, 1) = cumulants(6) - cumulants(1) * cumulants(1);
    covariance(2, 2) = cumulants(8) - cumulants(2) * cumulants(2);
    covariance(0, 1) = cumulants(4) - cumulants(0) * cumulants(1);
    covariance(1, 0) = covariance(0, 1);
    covariance(0, 2) = cumulants(5) - cumulants(0) * cumulants(2);
    covariance(2, 0) = covariance(0, 2);
    covariance(1, 2) = cumulants(7) - cumulants(1) * cumulants(2);
    covariance(2, 1) = covariance(1, 2);

    return FastEigen3x3(covariance);
}

struct compute_normal_functor {
    compute_normal_functor(const Eigen::Vector3f_u* points,
                           const int* indices, int knn)
        : points_(points), indices_(indices), knn_(knn) {};
    const Eigen::Vector3f_u* points_;
    const int* indices_;
    const int knn_;
    __device__
    Eigen::Vector3f_u operator()(const int& idx) const {
        KNNIndices idxs = KNNIndices::Constant(-1);
        for (int k = 0; k < knn_; ++k) idxs[k] = indices_[idx * knn_ + k];
        Eigen::Vector3f_u normal = ComputeNormal(points_, idxs, knn_);
        if (normal.norm() == 0.0) {
            normal = Eigen::Vector3f_u(0.0, 0.0, 1.0);
        }
        return normal;
    }
};

struct align_normals_direction {
    align_normals_direction(const Eigen::Vector3f& orientation_reference)
        : orientation_reference_(orientation_reference) {};
    const Eigen::Vector3f orientation_reference_;
    __device__
    void operator()(Eigen::Vector3f_u& normal) const {
        if (normal.norm() == 0.0) {
            normal = orientation_reference_;
        } else if (normal.dot(orientation_reference_) < 0.0) {
            normal *= -1.0;
        }
    }
};

}

bool PointCloud::EstimateNormals(const KDTreeSearchParam &search_param) {
    const bool has_normal = HasNormals();
    if (HasNormals() == false) {
        normals_.resize(points_.size());
    }
    KDTreeFlann kdtree;
    kdtree.SetGeometry(*this);
    thrust::device_vector<int> indices;
    thrust::device_vector<float> distance2;
    kdtree.Search(points_, search_param, indices, distance2);
    normals_.resize(points_.size());
    compute_normal_functor func(thrust::raw_pointer_cast(points_.data()),
                                thrust::raw_pointer_cast(indices.data()),
                                ((const KDTreeSearchParamKNN &)search_param).knn_);
    thrust::transform(indices.begin(), indices.end(), normals_.begin(), func);
    return true;
}

bool PointCloud::OrientNormalsToAlignWithDirection(const Eigen::Vector3f &orientation_reference) {
    if (HasNormals() == false) {
        utility::LogWarning(
                "[OrientNormalsToAlignWithDirection] No normals in the "
                "PointCloud. Call EstimateNormals() first.\n");
        return false;
    }
    align_normals_direction func(orientation_reference);
    thrust::for_each(normals_.begin(), normals_.end(), func);
    return true;
}
